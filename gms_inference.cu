#include "hip/hip_runtime.h"
#include <iostream>

#include "gms_fused_conv.cu"
#include "gms_fused_kernel.cu"
#include "gms_single_kernel.cu"

void Alex_Res_inference_thread(float *Alex_Layer1_Neurons,float *Alex_Layer2_Neurons,float *Alex_Layer3_Neurons,float *Alex_Layer4_Neurons,
					float *Alex_Layer5_Neurons,float *Alex_Layer6_Neurons,float *Alex_Layer7_Neurons,float *Alex_Layer8_Neurons,
                    float *Alex_Layer1_bias,float *Alex_Layer2_bias,float *Alex_Layer3_bias,float *Alex_Layer4_bias,
                    float *Alex_Layer5_bias,float *Alex_Layer6_bias,float *Alex_Layer7_bias,float *Alex_Layer8_bias,
                    float *Alex_Layer1_Weights,float *Alex_Layer2_Weights,float *Alex_Layer3_Weights,float *Alex_Layer4_Weights,
                    float *Alex_Layer5_Weights,float * Alex_Layer6_Weights,float *Alex_Layer7_Weights,float *Alex_Layer8_Weights,
                    float *Alex_Layer1_pool,float *Alex_Layer2_pool,float *Alex_Layer5_pool,
					float *Alex_Layer1_norm,float *Alex_Layer2_norm,float *Alex_Result_Neurons,
					float *Res_Layer1_Neurons,float *Res_Layer2_Neurons,float *Res_Layer3_Neurons,float *Res_Layer4_Neurons,
					float *Res_Layer5_Neurons,float *Res_Layer6_Neurons,float *Res_Layer7_Neurons,float *Res_Layer8_Neurons,
					float *Res_Layer9_Neurons,float *Res_Layer10_Neurons,float *Res_Layer11_Neurons,float *Res_Layer12_Neurons,
					float *Res_Layer13_Neurons,float *Res_Layer14_Neurons,float *Res_Layer15_Neurons,float *Res_Layer16_Neurons,
					float *Res_Layer17_Neurons,float *Res_Layer18_Neurons,
                    float *Res_Layer1_Weights,float *Res_Layer2_Weights,float *Res_Layer3_Weights,float *Res_Layer4_Weights,
                    float *Res_Layer5_Weights,float *Res_Layer6_Weights,float *Res_Layer7_Weights,float *Res_Layer8_Weights,
                    float *Res_Layer9_Weights,float *Res_Layer10_Weights,float *Res_Layer11_Weights,float *Res_Layer12_Weights,
                    float *Res_Layer13_Weights,float *Res_Layer14_Weights,float *Res_Layer15_Weights,float *Res_Layer16_Weights,
                    float *Res_Layer17_Weights,float *Res_Block3_Weights,float *Res_Block4_Weights,float *Res_Block5_Weights,
                    float *Res_Layer1_Gamma,float *Res_Layer2_Gamma,float *Res_Layer3_Gamma,float *Res_Layer4_Gamma,
                    float *Res_Layer5_Gamma,float *Res_Layer6_Gamma,float *Res_Layer7_Gamma,float *Res_Layer8_Gamma,
                    float *Res_Layer9_Gamma,float *Res_Layer10_Gamma,float *Res_Layer11_Gamma,float *Res_Layer12_Gamma,
                    float *Res_Layer13_Gamma,float *Res_Layer14_Gamma,float *Res_Layer15_Gamma,float *Res_Layer16_Gamma,
                    float *Res_Layer17_Gamma,float *Res_Block3_Gamma,float *Res_Block4_Gamma,float *Res_Block5_Gamma,
                    float *Res_Layer1_Beta,float *Res_Layer2_Beta,float *Res_Layer3_Beta,float *Res_Layer4_Beta,
                    float *Res_Layer5_Beta,float *Res_Layer6_Beta,float *Res_Layer7_Beta,float *Res_Layer8_Beta,
                    float *Res_Layer9_Beta,float *Res_Layer10_Beta,float *Res_Layer11_Beta,float *Res_Layer12_Beta,
                    float *Res_Layer13_Beta,float *Res_Layer14_Beta,float *Res_Layer15_Beta,float *Res_Layer16_Beta,
                    float *Res_Layer17_Beta,float *Res_Block3_Beta,float *Res_Block4_Beta,float *Res_Block5_Beta,
                    float *Res_mean1,float *Res_mean2,float *Res_mean3,float *Res_mean4,float *Res_mean5,
                    float *Res_mean6,float *Res_mean7,float *Res_mean8,float *Res_mean9,float *Res_mean10,
                    float *Res_mean11,float *Res_mean12,float *Res_mean13,float *Res_mean14,float *Res_mean15,
                    float *Res_mean16,float *Res_mean17,float *Res_Block3_mean,float *Res_Block4_mean,float *Res_Block5_mean,
                    float *Res_var1,float *Res_var2,float *Res_var3,float *Res_var4,float *Res_var5,
                    float *Res_var6,float *Res_var7,float *Res_var8,float *Res_var9,float *Res_var10,
                    float *Res_var11,float *Res_var12,float *Res_var13,float *Res_var14,float *Res_var15,
                    float *Res_var16,float *Res_var17,float *Res_Block3_var,float *Res_Block4_var,float *Res_Block5_var,
                    float *Res_FC_bias,float *Res_FC_Weights,
					float *Res_Layer3_basic,float *Res_Layer5_basic,float *Res_Layer7_basic,float *Res_Layer9_basic,
					float *Res_Layer11_basic,float *Res_Layer13_basic,float *Res_Layer15_basic,float *Res_Layer17_basic,
					float *Res_Block3_basic,float *Res_Block4_basic,float *Res_Block5_basic,
					float *Res_Layer1_bn,float *Res_Layer2_bn,float *Res_Layer3_bn,float *Res_Layer4_bn,
					float *Res_Layer5_bn,float *Res_Layer6_bn,float *Res_Layer7_bn,float *Res_Layer8_bn,
					float *Res_Layer9_bn,float *Res_Layer10_bn,float *Res_Layer11_bn,float *Res_Layer12_bn,
					float *Res_Layer13_bn,float *Res_Layer14_bn,float *Res_Layer15_bn,float *Res_Layer16_bn,
					float *Res_Layer17_bn,float *Res_Block3_bn,float *Res_Block4_bn,float *Res_Block5_bn,
					float *Res_Layer1_pool,float *Res_FC_Neurons,float *Res_Result_Neurons,
					int alex_num, int res_num,
					int gridSize, int alexWarpNum, int resWarpNum)
{
    // int gridSize = 0;
    int blockSize = max(alexWarpNum, resWarpNum);
    int alexNumOps = 0;
    int resNumOps = 0;

    int a_in_fm, r_in_fm, a_in_channel, r_in_channel;
    int a_out_fm, r_out_fm, a_out_channel, r_out_channel;
    int a_ker_fm, r_ker_fm;
    int a_str, r_str;
    int a_pad, r_pad;
    bool a_relu, r_relu;

    /* Fusing First convolution */
	// dim3 Block1(64,9,9);
	// dim3 Thread1(28,28,1);
	// fused_first_layer<<<Block1,Thread1>>>(Alex_Layer1_bias,Alex_Layer1_Weights,Res_Layer1_Weights,
	// 									Alex_Layer1_Neurons,Res_Layer1_Neurons,
	// 									Alex_Layer1_norm,Res_Layer1_bn,
	// 									alex_num,res_num,
	// 									224,55,4,2,11,3,
	// 									224,112,2,3,7,3,
	// 									5,11,
	// 									4,28);

    // gridSize = 264;
    dim3 Block1(gridSize,1,1);
    dim3 Thread1(32,blockSize,1);

    a_in_fm         = 224;  a_in_channel    = 3;
    a_out_fm        = 55;   a_out_channel   = 64;
    a_ker_fm        = 11;
    a_str           = 4;    a_pad           = 2;    a_relu          = true;

    r_in_fm         = 224;  r_in_channel    = 3;
    r_out_fm        = 112;  r_out_channel   = 64;
    r_ker_fm        = 7;    
    r_str           = 2;    r_pad           = 3;    r_relu          = true;

    alexNumOps = (a_out_fm*a_out_fm*a_out_channel - 1) / (gridSize * 32 * alexWarpNum) + 1;
    resNumOps = (r_out_fm*r_out_fm*r_out_channel - 1) / (gridSize * 32 * resWarpNum) + 1;

    // fused_first_conv_thread<<<Block1, Thread1>>>(Alex_Layer1_bias,Alex_Layer1_Weights,Res_Layer1_Weights,
    //                                 Alex_Layer1_Neurons,Res_Layer1_Neurons,
    //                                 Alex_Layer1_norm,Res_Layer1_bn,
    //                                 alex_num,res_num,
    //                                 a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
    //                                 r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
    //                                 alexWarpNum,resWarpNum,
    //                                 alexNumOps,resNumOps);

    /* Alex 1st lrm + Res 1st bn */
	dim3 Block2(64,8,8);
    dim3 Thread2(14,14,1);
    fused_lrm_bn1<<<Block2,Thread2>>>(Alex_Layer1_norm,Res_Layer1_bn,
									Alex_Layer1_pool,Res_Layer1_pool,
									alex_num,res_num,
									0.0001,0.75,5,55,
									Res_mean1,Res_var1,Res_Layer1_Gamma,Res_Layer1_Beta,112,true,
									64,5,11,
									64,8,14);

    /* Alex 1st max + Res 1st max */
    dim3 Block3(64,7,7);
    dim3 Thread3(9,9);
    fused_max1<<<Block3,Thread3>>>(Alex_Layer1_pool,Res_Layer1_pool,
                                    Alex_Layer2_Neurons,Res_Layer2_Neurons,
                                    alex_num,res_num,
                                    55,27,2,0,3,
                                    112,56,2,1,3,
                                    64,3,9,
                                    64,7,8);

    /* Alex 2nd conv + Res 2nd conv */
	/* Original */
	// dim3 Block4(192,8,8);
    // dim3 Thread4(9,9,1);
	// fused_two_conv<<<Block4,Thread4>>>(Alex_Layer2_bias,Alex_Layer2_Weights,Res_Layer2_Weights,
    //                                     Alex_Layer2_Neurons,Res_Layer2_Neurons,
    //                                     Alex_Layer2_norm,Res_Layer2_bn,
    //                                     alex_num,res_num,
    //                                     27,27,1,2,5,64,true,
    //                                     56,56,1,1,3,64,false,
    //                                     192,3,9,
    //                                     64,8,7);
	
    /*** test_conv_inner_thread ***/
    dim3 Block4(gridSize,1,1);
    dim3 Thread4(32,blockSize,1);

    a_in_fm         = 27;   a_in_channel    = 64;
    a_out_fm        = 27;   a_out_channel   = 192;
    a_ker_fm        = 5;
    a_str           = 1;    a_pad           = 2;    a_relu          = true;

    r_in_fm         = 56;   r_in_channel    = 64;
    r_out_fm        = 56;   r_out_channel   = 64;
    r_ker_fm        = 3;    
    r_str           = 1;    r_pad           = 1;    r_relu          = false;

    alexNumOps = (a_out_fm*a_out_fm*a_out_channel - 1) / (gridSize * 32 * alexWarpNum) + 1;
    resNumOps = (r_out_fm*r_out_fm*r_out_channel - 1) / (gridSize * 32 * resWarpNum) + 1;

    fused_two_conv_thread<<<Block4, Thread4>>>(Alex_Layer2_bias,Alex_Layer2_Weights,Res_Layer2_Weights,
                                    Alex_Layer2_Neurons,Res_Layer2_Neurons,
                                    Alex_Layer2_norm,Res_Layer2_bn,
                                    alex_num,res_num,
                                    a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
                                    r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
                                    alexWarpNum,resWarpNum,
                                    alexNumOps,resNumOps);


    /* Alex 2nd lrm + Res 2nd bn */
    dim3 Block5(192,2,2);
    dim3 Thread5(28,28);
    fused_lrm_bn1<<<Block5,Thread5>>>(Alex_Layer2_norm,Res_Layer2_bn,
                                    Alex_Layer2_pool,Res_Layer3_Neurons,
                                    alex_num,res_num,
                                    0.0001,0.75,5,27,
                                    Res_mean2,Res_var2,Res_Layer2_Gamma,Res_Layer2_Beta,56,true,
                                    192,1,27,
                                    64,2,28);


    /* Alex 2nd max */
    dim3 Block6(192,1,1);
    dim3 Thread6(13,13);
	max_jjb<<<Block6,Thread6>>>(Alex_Layer2_pool,Alex_Layer3_Neurons,alex_num,27,13,2,0,3);


    /* Alex 3rd conv + Res 3rd conv ********************************************************************************************************************/   
    // dim3 Block7(64,4,4);
    // dim3 Thread7(14,14,1);
	// fused_two_conv<<<Block7,Thread7>>>(Alex_Layer3_bias,Alex_Layer3_Weights,Res_Layer3_Weights,
    //                                     Alex_Layer3_Neurons,Res_Layer3_Neurons,
    //                                     Alex_Layer4_Neurons,Res_Layer3_bn,
    //                                     alex_num,res_num,
    //                                     13,13,1,1,3,192,true,
    //                                     56,56,1,1,3,64,false,
    //                                     64,1,13,
    //                                     64,4,14);

    /*** test_conv_inner_thread ***/
    dim3 Block7(gridSize,1,1);
    dim3 Thread7(32,blockSize,1);

    a_in_fm         = 13;   a_in_channel    = 192;
    a_out_fm        = 13;   a_out_channel   = 64;
    a_ker_fm        = 3;
    a_str           = 1;    a_pad           = 1;    a_relu          = true;

    r_in_fm         = 56;   r_in_channel    = 64;
    r_out_fm        = 56;   r_out_channel   = 64;
    r_ker_fm        = 3;    
    r_str           = 1;    r_pad           = 1;    r_relu          = false;
    
    alexNumOps = (a_out_fm*a_out_fm*a_out_channel - 1) / (gridSize * 32 * alexWarpNum) + 1;
    resNumOps = (r_out_fm*r_out_fm*r_out_channel - 1) / (gridSize * 32 * resWarpNum) + 1;

    fused_two_conv_thread<<<Block7, Thread7>>>(Alex_Layer3_bias,Alex_Layer3_Weights,Res_Layer3_Weights,
                                    Alex_Layer3_Neurons,Res_Layer3_Neurons,
                                    Alex_Layer4_Neurons,Res_Layer3_bn,
                                    alex_num,res_num,
                                    a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
                                    r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
                                    alexWarpNum,resWarpNum,
                                    alexNumOps,resNumOps);

	// dim3 Block7_1(320,1,1);
    // dim3 Thread7_1(13,13,1);
	// conv_jjb1<<<Block7_1,Thread7_1>>>(Alex_Layer3_bias,Alex_Layer3_Neurons,Alex_Layer3_Weights,Alex_Layer4_Neurons,alex_num,13,13,1,1,3,192,true,true);

    /* Res 3rd bn */
    dim3 Block8(64,8,8);
    dim3 Thread8(7,7);
 	batchnorm_jjb<<<Block8,Thread8>>>(Res_Layer3_bn,Res_Layer3_basic,res_num,Res_mean3,Res_var3,Res_Layer3_Gamma,Res_Layer3_Beta,56,false);
   
    /* Res 3rd basic */
    dim3 Block9(64,8,8);
    dim3 Thread9(7,7);
    basic_block_jjb<<<Block9,Thread9>>>(Res_Layer2_Neurons,Res_Layer3_basic,Res_Layer4_Neurons,res_num,56,true);


    /* Alex 4th conv + Res 4th conv */
    dim3 Block10(64,4,4);
    dim3 Thread10(14,14);
	fused_two_conv<<<Block10,Thread10>>>(Alex_Layer4_bias,Alex_Layer4_Weights,Res_Layer4_Weights,
                                            Alex_Layer4_Neurons,Res_Layer4_Neurons,
                                            Alex_Layer5_Neurons,Res_Layer4_bn,
                                            alex_num,res_num,
                                            13,13,1,1,3,384,true,
                                            56,56,1,1,3,64,false,
                                            64,1,13,
                                            64,4,14);
	// dim3 Block10_1(192,1,1);
    // dim3 Thread10_1(13,13,1);
	// conv_jjb1<<<Block10_1,Thread10_1>>>(Alex_Layer4_bias,Alex_Layer4_Neurons,Alex_Layer4_Weights,Alex_Layer5_Neurons,alex_num,13,13,1,1,3,384,true,true);

    /* Res 4th bn */
    dim3 Block11(64,7,7);
    dim3 Thread11(8,8);
	batchnorm_jjb<<<Block11,Thread11>>>(Res_Layer4_bn,Res_Layer5_Neurons,res_num,Res_mean4,Res_var4,Res_Layer4_Gamma,Res_Layer4_Beta,56,true);
	
    /* Alex 5th conv + Res 5th conv */
    dim3 Block12(64,4,4);
    dim3 Thread12(14,14);
	fused_two_conv<<<Block12,Thread12>>>(Alex_Layer5_bias,Alex_Layer5_Weights,Res_Layer5_Weights,
                                            Alex_Layer5_Neurons,Res_Layer5_Neurons,
                                            Alex_Layer5_pool,Res_Layer5_bn,
                                            alex_num,res_num,
                                            13,13,1,1,3,256,true,
                                            56,56,1,1,3,64,false,
                                            64,1,13,
                                            64,4,14);
	// dim3 Block12_1(192,1,1);
    // dim3 Thread12_1(13,13,1);									
	// conv_jjb1<<<Block12_1,Thread12_1>>>(Alex_Layer5_bias,Alex_Layer5_Neurons,Alex_Layer5_Weights,Alex_Layer5_pool,alex_num,13,13,1,1,3,256,true,true);
										
    /* Alex 5th max + Res 5th bn */
	dim3 Block13(256,7,7);
	dim3 Thread13(8,8);
	fused_bn_max1<<<Block13,Thread13>>>(Res_Layer5_bn,Alex_Layer5_pool,
	                                    Res_Layer5_basic,Alex_Layer6_Neurons,
	                                    res_num,alex_num,
	                                    Res_mean5,Res_var5,Res_Layer5_Gamma,Res_Layer5_Beta,56,false,
	                                    13,6,2,0,3,
										64,7,8,
	                                    256,1,6);


    /* Res 5th basic */
	dim3 Block14(64,8,8);
    dim3 Thread14(7,7);
    basic_block_jjb<<<Block14,Thread14>>>(Res_Layer4_Neurons,Res_Layer5_basic,Res_Layer6_Neurons,res_num,56,true);

   	//6th layer
	dim3 Block15(128,4,4);
    dim3 Thread15(7,7);
	conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer6_Neurons,Res_Layer6_Weights,Res_Layer6_bn,res_num,56,28,2,1,3,64,false,false);
	batchnorm_jjb<<<Block15,Thread15>>>(Res_Layer6_bn,Res_Layer7_Neurons,res_num,Res_mean6,Res_var6,Res_Layer6_Gamma,Res_Layer6_Beta,28,true);

	//7th layer
	conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer7_Neurons,Res_Layer7_Weights,Res_Layer7_bn,res_num,28,28,1,1,3,128,false,false);
	batchnorm_jjb<<<Block15,Thread15>>>(Res_Layer7_bn,Res_Layer7_basic,res_num,Res_mean7,Res_var7,Res_Layer7_Gamma,Res_Layer7_Beta,28,false);

	//Block B output
	conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer6_Neurons,Res_Block3_Weights,Res_Block3_bn,res_num,56,28,2,0,1,64,false,false); 
	batchnorm_jjb<<<Block15,Thread15>>>(Res_Block3_bn,Res_Block3_basic,res_num,Res_Block3_mean,Res_Block3_var,Res_Block3_Gamma,Res_Block3_Beta,28,false);

	basic_block_jjb<<<Block15,Thread15>>>(Res_Layer7_basic,Res_Block3_basic,Res_Layer8_Neurons,res_num,28,true);

	//8th layer
	conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer8_Neurons,Res_Layer8_Weights,Res_Layer8_bn,res_num,28,28,1,1,3,128,false,false);
	batchnorm_jjb<<<Block15,Thread15>>>(Res_Layer8_bn,Res_Layer9_Neurons,res_num,Res_mean8,Res_var8,Res_Layer8_Gamma,Res_Layer8_Beta,28,true);

	//9th layer
	conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer9_Neurons,Res_Layer9_Weights,Res_Layer9_bn,res_num,28,28,1,1,3,128,false,false);
	batchnorm_jjb<<<Block15,Thread15>>>(Res_Layer9_bn,Res_Layer9_basic,res_num,Res_mean9,Res_var9,Res_Layer9_Gamma,Res_Layer9_Beta,28,false);

	basic_block_jjb<<<Block15,Thread15>>>(Res_Layer8_Neurons,Res_Layer9_basic,Res_Layer10_Neurons,res_num,28,true);

    /* Res 10th conv */
    dim3 Block27(256,2,2);
    dim3 Thread27(7,7);
	conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer10_Neurons,Res_Layer10_Weights,Res_Layer10_bn,res_num,28,14,2,1,3,128,false,false);

    //10th layer
	conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer10_Neurons,Res_Layer10_Weights,Res_Layer10_bn,res_num,28,14,2,1,3,128,false,false);
	batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer10_bn,Res_Layer11_Neurons,res_num,Res_mean10,Res_var10,Res_Layer10_Gamma,Res_Layer10_Beta,14,true);

	//11th layer
	conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer11_Neurons,Res_Layer11_Weights,Res_Layer11_bn,res_num,14,14,1,1,3,256,false,false);
	batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer11_bn,Res_Layer11_basic,res_num,Res_mean11,Res_var11,Res_Layer11_Gamma,Res_Layer11_Beta,14,false);

    /* Res 11th bn */
	batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer11_bn,Res_Layer11_basic,res_num,Res_mean11,Res_var11,Res_Layer11_Gamma,Res_Layer11_Beta,14,false);

    /* Res 11th block conv + bn + basic */
	conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer10_Neurons,Res_Block4_Weights,Res_Block4_bn,res_num,28,14,2,0,1,128,false,false);
	batchnorm_jjb<<<Block27,Thread27>>>(Res_Block4_bn,Res_Block4_basic,res_num,Res_Block4_mean,Res_Block4_var,Res_Block4_Gamma,Res_Block4_Beta,14,false);
	basic_block_jjb<<<Block27,Thread27>>>(Res_Layer11_basic,Res_Block4_basic,Res_Layer12_Neurons,res_num,14,true);

	//12th layer
	conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer12_Neurons,Res_Layer12_Weights,Res_Layer12_bn,res_num,14,14,1,1,3,256,false,false);
	batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer12_bn,Res_Layer13_Neurons,res_num,Res_mean12,Res_var12,Res_Layer12_Gamma,Res_Layer12_Beta,14,true);

	//13th layer
	conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer13_Neurons,Res_Layer13_Weights,Res_Layer13_bn,res_num,14,14,1,1,3,256,false,false); 
	batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer13_bn,Res_Layer13_basic,res_num,Res_mean13,Res_var13,Res_Layer13_Gamma,Res_Layer13_Beta,14,false);

	basic_block_jjb<<<Block27,Thread27>>>(Res_Layer12_Neurons,Res_Layer13_basic,Res_Layer14_Neurons,res_num,14,true);

    /* Res 14th ~ 17th + 18th avgpooling*/
    dim3 Block39(512,1,1);
    dim3 Thread39(7,7);
    // Res 14th 
	conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer14_Neurons,Res_Layer14_Weights,Res_Layer14_bn,res_num,14,7,2,1,3,256,false,false);
	batchnorm_jjb<<<Block39,Thread39>>>(Res_Layer14_bn,Res_Layer15_Neurons,res_num,Res_mean14,Res_var14,Res_Layer14_Gamma,Res_Layer14_Beta,7,true);

    // Res 15th
	conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer15_Neurons,Res_Layer15_Weights,Res_Layer15_bn,res_num,7,7,1,1,3,512,false,false);
	batchnorm_jjb<<<Block39,Thread39>>>(Res_Layer15_bn,Res_Layer15_basic,res_num,Res_mean15,Res_var15,Res_Layer15_Gamma,Res_Layer15_Beta,7,false);

	//Block D output
	conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer14_Neurons,Res_Block5_Weights,Res_Block5_bn,res_num,14,7,2,0,1,256,false,false);
	batchnorm_jjb<<<Block39,Thread39>>>(Res_Block5_bn,Res_Block5_basic,res_num,Res_Block5_mean,Res_Block5_var,Res_Block5_Gamma,Res_Block5_Beta,7,false);
	basic_block_jjb<<<Block39,Thread39>>>(Res_Layer15_basic,Res_Block5_basic,Res_Layer16_Neurons,res_num,7,true);

    // Res 16th
	conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer16_Neurons,Res_Layer16_Weights,Res_Layer16_bn,res_num,7,7,1,1,3,512,false,false);
	batchnorm_jjb<<<Block39,Thread39>>>(Res_Layer16_bn,Res_Layer17_Neurons,res_num,Res_mean16,Res_var16,Res_Layer16_Gamma,Res_Layer16_Beta,7,true);
	
    // Res 17th
	conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer17_Neurons,Res_Layer17_Weights,Res_Layer17_bn,res_num,7,7,1,1,3,512,false,false); 
	batchnorm_jjb<<<Block39,Thread39>>>(Res_Layer17_bn,Res_Layer17_basic,res_num,Res_mean17,Res_var17,Res_Layer17_Gamma,Res_Layer17_Beta,7,false);

	basic_block_jjb<<<Block39,Thread39>>>(Res_Layer16_Neurons,Res_Layer17_basic,Res_Layer18_Neurons,res_num,7,true);

    // Res 18th avgpooling
    dim3 Block40(512,1,1);
    dim3 Thread40(1,1);
	globalavg_jjb<<<Block40,Thread40>>>(Res_Layer18_Neurons,Res_FC_Neurons,res_num,7);

    /* Alex 6th fc */
    dim3 block41(4096,1,1);
    dim3 Thread41(1,1);

	fc_jjb<<<block41,Thread41>>>(Alex_Layer6_bias,Alex_Layer6_Neurons,Alex_Layer6_Weights,Alex_Layer7_Neurons,alex_num,(6*6*256),true);
    
	/* Alex 7th fc */
    dim3 block42(4096,1,1);
    dim3 Thread42(1,1);

	fc_jjb<<<block42,Thread42>>>(Alex_Layer7_bias,Alex_Layer7_Neurons,Alex_Layer7_Weights,Alex_Layer8_Neurons,alex_num,4096,true);

    /* Alex 8th fc + Res 18th fc */
    dim3 block43(1000,1,1);
    dim3 Thread43(1,1);
    fused_two_fc1<<<block43,Thread43>>>(Alex_Layer8_bias,Res_FC_bias,Alex_Layer8_Weights,Res_FC_Weights,
                                        Alex_Layer8_Neurons,Res_FC_Neurons,
                                        Alex_Result_Neurons,Res_Result_Neurons,
                                        alex_num,res_num,
                                        4096, false,
		                                512,false);



    for(int j = 0; j < alex_num; j++){
        float *Alex_Result_Neurons_CPU = (float *) malloc ((1000) * sizeof(float));
		hipMemcpy(Alex_Result_Neurons_CPU, Alex_Result_Neurons, (1000) * sizeof(float), hipMemcpyDeviceToHost);

		float max_alex = 0.0;
		int index_alex = 0;
		for(int i = 0; i < 1000; i++){
			if(max_alex < Alex_Result_Neurons_CPU[i]){
				max_alex = Alex_Result_Neurons_CPU[i];	
				index_alex = i;
			}
		}

		int line_count_alex = 0;
        char buffer_alex[1000];
        FILE *list_alex = fopen("imagenet1000_clsidx_to_labels.txt","rt");
        while(fgets(buffer_alex, 1000, list_alex) != NULL){
            line_count_alex++;
            if(line_count_alex == (index_alex+1)){
                printf("%f Alex: %s", max_alex, buffer_alex);
                // if (strcmp(buffer_alex, "Egyptian cat") != 0)
                // {
                //     printf("\n---Alexnet Result---");
                //     printf("\nClass ID: %d\nClass Name: %sProbability: %.20f\n\n", index_alex, buffer_alex, max_alex);
                //     exit(1);
                // }
                // printf("Alexnet: %d, %s", index_alex, buffer_alex);
                break;
            }
        }
        fclose(list_alex);
		// free(Alex_Result_Neurons_CPU);
    }


	for(int j = 0; j < res_num; j++){
        float *Res_Result_Neurons_CPU = (float *) malloc ((1000) * sizeof(float));
		hipMemcpy(Res_Result_Neurons_CPU, Res_Result_Neurons, (1000) * sizeof(float), hipMemcpyDeviceToHost);

		float max_res = 0.0;
        int index_res = 0; 
        for(int i = 0; i < 1000; i++){
            if(max_res < Res_Result_Neurons_CPU[i]){
                max_res = Res_Result_Neurons_CPU[i];	
                index_res = i;
            }
        }	
        int line_count_res = 0;
        char buffer_res[1000];
        FILE *list_res = fopen("imagenet1000_clsidx_to_labels.txt","rt");
        while(fgets(buffer_res, 1000, list_res) != NULL){
            line_count_res++;
            if(line_count_res == (index_res+1)){
                printf("%f Res: %s", max_res, buffer_res);
                // if (strcmp(buffer_res, "tabby, tabby cat") != 0)
                // {
                //     printf("\n---Resnet18 Result---");
                //     printf("\nClass ID: %d\nClass Name: %sProbability: %.20f\n\n", index_res, buffer_res, max_res);
                //     exit(1);
                // }
                // printf("Resnet18: %d, %s", index_res, buffer_res);
                break;
            }
        }
        fclose(list_res);
		// free(Res_Result_Neurons_CPU);
    }
}

void Alex_Res_inference_block(float *Alex_Layer1_Neurons,float *Alex_Layer2_Neurons,float *Alex_Layer3_Neurons,float *Alex_Layer4_Neurons,
					float *Alex_Layer5_Neurons,float *Alex_Layer6_Neurons,float *Alex_Layer7_Neurons,float *Alex_Layer8_Neurons,
                    float *Alex_Layer1_bias,float *Alex_Layer2_bias,float *Alex_Layer3_bias,float *Alex_Layer4_bias,
                    float *Alex_Layer5_bias,float *Alex_Layer6_bias,float *Alex_Layer7_bias,float *Alex_Layer8_bias,
                    float *Alex_Layer1_Weights,float *Alex_Layer2_Weights,float *Alex_Layer3_Weights,float *Alex_Layer4_Weights,
                    float *Alex_Layer5_Weights,float * Alex_Layer6_Weights,float *Alex_Layer7_Weights,float *Alex_Layer8_Weights,
                    float *Alex_Layer1_pool,float *Alex_Layer2_pool,float *Alex_Layer5_pool,
					float *Alex_Layer1_norm,float *Alex_Layer2_norm,float *Alex_Result_Neurons,
					float *Res_Layer1_Neurons,float *Res_Layer2_Neurons,float *Res_Layer3_Neurons,float *Res_Layer4_Neurons,
					float *Res_Layer5_Neurons,float *Res_Layer6_Neurons,float *Res_Layer7_Neurons,float *Res_Layer8_Neurons,
					float *Res_Layer9_Neurons,float *Res_Layer10_Neurons,float *Res_Layer11_Neurons,float *Res_Layer12_Neurons,
					float *Res_Layer13_Neurons,float *Res_Layer14_Neurons,float *Res_Layer15_Neurons,float *Res_Layer16_Neurons,
					float *Res_Layer17_Neurons,float *Res_Layer18_Neurons,
                    float *Res_Layer1_Weights,float *Res_Layer2_Weights,float *Res_Layer3_Weights,float *Res_Layer4_Weights,
                    float *Res_Layer5_Weights,float *Res_Layer6_Weights,float *Res_Layer7_Weights,float *Res_Layer8_Weights,
                    float *Res_Layer9_Weights,float *Res_Layer10_Weights,float *Res_Layer11_Weights,float *Res_Layer12_Weights,
                    float *Res_Layer13_Weights,float *Res_Layer14_Weights,float *Res_Layer15_Weights,float *Res_Layer16_Weights,
                    float *Res_Layer17_Weights,float *Res_Block3_Weights,float *Res_Block4_Weights,float *Res_Block5_Weights,
                    float *Res_Layer1_Gamma,float *Res_Layer2_Gamma,float *Res_Layer3_Gamma,float *Res_Layer4_Gamma,
                    float *Res_Layer5_Gamma,float *Res_Layer6_Gamma,float *Res_Layer7_Gamma,float *Res_Layer8_Gamma,
                    float *Res_Layer9_Gamma,float *Res_Layer10_Gamma,float *Res_Layer11_Gamma,float *Res_Layer12_Gamma,
                    float *Res_Layer13_Gamma,float *Res_Layer14_Gamma,float *Res_Layer15_Gamma,float *Res_Layer16_Gamma,
                    float *Res_Layer17_Gamma,float *Res_Block3_Gamma,float *Res_Block4_Gamma,float *Res_Block5_Gamma,
                    float *Res_Layer1_Beta,float *Res_Layer2_Beta,float *Res_Layer3_Beta,float *Res_Layer4_Beta,
                    float *Res_Layer5_Beta,float *Res_Layer6_Beta,float *Res_Layer7_Beta,float *Res_Layer8_Beta,
                    float *Res_Layer9_Beta,float *Res_Layer10_Beta,float *Res_Layer11_Beta,float *Res_Layer12_Beta,
                    float *Res_Layer13_Beta,float *Res_Layer14_Beta,float *Res_Layer15_Beta,float *Res_Layer16_Beta,
                    float *Res_Layer17_Beta,float *Res_Block3_Beta,float *Res_Block4_Beta,float *Res_Block5_Beta,
                    float *Res_mean1,float *Res_mean2,float *Res_mean3,float *Res_mean4,float *Res_mean5,
                    float *Res_mean6,float *Res_mean7,float *Res_mean8,float *Res_mean9,float *Res_mean10,
                    float *Res_mean11,float *Res_mean12,float *Res_mean13,float *Res_mean14,float *Res_mean15,
                    float *Res_mean16,float *Res_mean17,float *Res_Block3_mean,float *Res_Block4_mean,float *Res_Block5_mean,
                    float *Res_var1,float *Res_var2,float *Res_var3,float *Res_var4,float *Res_var5,
                    float *Res_var6,float *Res_var7,float *Res_var8,float *Res_var9,float *Res_var10,
                    float *Res_var11,float *Res_var12,float *Res_var13,float *Res_var14,float *Res_var15,
                    float *Res_var16,float *Res_var17,float *Res_Block3_var,float *Res_Block4_var,float *Res_Block5_var,
                    float *Res_FC_bias,float *Res_FC_Weights,
					float *Res_Layer3_basic,float *Res_Layer5_basic,float *Res_Layer7_basic,float *Res_Layer9_basic,
					float *Res_Layer11_basic,float *Res_Layer13_basic,float *Res_Layer15_basic,float *Res_Layer17_basic,
					float *Res_Block3_basic,float *Res_Block4_basic,float *Res_Block5_basic,
					float *Res_Layer1_bn,float *Res_Layer2_bn,float *Res_Layer3_bn,float *Res_Layer4_bn,
					float *Res_Layer5_bn,float *Res_Layer6_bn,float *Res_Layer7_bn,float *Res_Layer8_bn,
					float *Res_Layer9_bn,float *Res_Layer10_bn,float *Res_Layer11_bn,float *Res_Layer12_bn,
					float *Res_Layer13_bn,float *Res_Layer14_bn,float *Res_Layer15_bn,float *Res_Layer16_bn,
					float *Res_Layer17_bn,float *Res_Block3_bn,float *Res_Block4_bn,float *Res_Block5_bn,
					float *Res_Layer1_pool,float *Res_FC_Neurons,float *Res_Result_Neurons,
					int alex_num, int res_num,
                    int gridSize, int alexWarpNum, int resWarpNum) 
{
    // int gridSize = 0;
    int blockSize = alexWarpNum + resWarpNum;
    int alexNumOps = 0;
    int resNumOps = 0;

    int a_in_fm, r_in_fm, a_in_channel, r_in_channel;
    int a_out_fm, r_out_fm, a_out_channel, r_out_channel;
    int a_ker_fm, r_ker_fm;
    int a_str, r_str;
    int a_pad, r_pad;
    bool a_relu, r_relu;

    /*** Fusing First convolution ***/
    // gridSize = 264;
    dim3 Block1(gridSize,1,1);
    dim3 Thread1(32,blockSize,1);

    a_in_fm         = 224;  a_in_channel    = 3;
    a_out_fm        = 55;   a_out_channel   = 64;
    a_ker_fm        = 11;
    a_str           = 4;    a_pad           = 2;    a_relu          = true;

    r_in_fm         = 224;  r_in_channel    = 3;
    r_out_fm        = 112;  r_out_channel   = 64;
    r_ker_fm        = 7;    
    r_str           = 2;    r_pad           = 3;    r_relu          = true;

    alexNumOps = (a_out_fm*a_out_fm*a_out_channel - 1) / (gridSize * 32 * alexWarpNum) + 1;
    resNumOps = (r_out_fm*r_out_fm*r_out_channel - 1) / (gridSize * 32 * resWarpNum) + 1;

    fused_two_conv_block<<<Block1, Thread1>>>(Alex_Layer1_bias,Alex_Layer1_Weights,Res_Layer1_Weights,
                                    Alex_Layer1_Neurons,Res_Layer1_Neurons,
                                    Alex_Layer1_norm,Res_Layer1_bn,
                                    alex_num,res_num,
                                    a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
                                    r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
                                    alexWarpNum,resWarpNum,
                                    alexNumOps,resNumOps,
                                    true);

    // fused_two_conv_block_sh<<<Block1, Thread1>>>(Alex_Layer1_bias,Alex_Layer1_Weights,Res_Layer1_Weights,
    //                                 Alex_Layer1_Neurons,Res_Layer1_Neurons,
    //                                 Alex_Layer1_norm,Res_Layer1_bn,
    //                                 alex_num,res_num,
    //                                 a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
    //                                 r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
    //                                 alexWarpNum,resWarpNum,
    //                                 alexNumOps,resNumOps,
    //                                 100,0,
    //                                 true);

    // hipError_t err{hipGetLastError()};
    // std::cerr << hipGetErrorString(err) << std::endl;

    /* Alex 1st lrm + Res 1st bn */
	dim3 Block2(64,8,8);
    dim3 Thread2(14,14,1);
    fused_lrm_bn1<<<Block2,Thread2>>>(Alex_Layer1_norm,Res_Layer1_bn,
									Alex_Layer1_pool,Res_Layer1_pool,
									alex_num,res_num,
									0.0001,0.75,5,55,
									Res_mean1,Res_var1,Res_Layer1_Gamma,Res_Layer1_Beta,112,true,
									64,5,11,
									64,8,14);

    /* Alex 1st max + Res 1st max */
    dim3 Block3(64,7,7);
    dim3 Thread3(9,9);
    fused_max1<<<Block3,Thread3>>>(Alex_Layer1_pool,Res_Layer1_pool,
                                    Alex_Layer2_Neurons,Res_Layer2_Neurons,
                                    alex_num,res_num,
                                    55,27,2,0,3,
                                    112,56,2,1,3,
                                    64,3,9,
                                    64,7,8);

    /************* Alex 2nd conv + Res 2nd conv *********************************************************************************************************/
	/*** Alex 2rd conv + Res 2rd conv ***/
    // gridSize = 4384;
    dim3 Block4(gridSize,1,1);
    dim3 Thread4(32,blockSize,1);

    a_in_fm         = 27;   a_in_channel    = 64;
    a_out_fm        = 27;   a_out_channel   = 192;
    a_ker_fm        = 5;
    a_str           = 1;    a_pad           = 2;    a_relu          = true;

    r_in_fm         = 56;   r_in_channel    = 64;
    r_out_fm        = 56;   r_out_channel   = 64;
    r_ker_fm        = 3;    
    r_str           = 1;    r_pad           = 1;    r_relu          = false;

    alexNumOps = (a_out_fm*a_out_fm*a_out_channel - 1) / (gridSize * 32 * alexWarpNum) + 1;
    resNumOps = (r_out_fm*r_out_fm*r_out_channel - 1) / (gridSize * 32 * resWarpNum) + 1;

    fused_two_conv_block<<<Block4, Thread4>>>(Alex_Layer2_bias,Alex_Layer2_Weights,Res_Layer2_Weights,
                                    Alex_Layer2_Neurons,Res_Layer2_Neurons,
                                    Alex_Layer2_norm,Res_Layer2_bn,
                                    alex_num,res_num,
                                    a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
                                    r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
                                    alexWarpNum,resWarpNum,
                                    alexNumOps,resNumOps,
                                    false);

    /* Alex 2nd lrm + Res 2nd bn */
    dim3 Block5(192,2,2);
    dim3 Thread5(28,28);
    fused_lrm_bn1<<<Block5,Thread5>>>(Alex_Layer2_norm,Res_Layer2_bn,
                                    Alex_Layer2_pool,Res_Layer3_Neurons,
                                    alex_num,res_num,
                                    0.0001,0.75,5,27,
                                    Res_mean2,Res_var2,Res_Layer2_Gamma,Res_Layer2_Beta,56,true,
                                    192,1,27,
                                    64,2,28);


    /* Alex 2nd max */
    dim3 Block6(192,1,1);
    dim3 Thread6(13,13);
	max_jjb<<<Block6,Thread6>>>(Alex_Layer2_pool,Alex_Layer3_Neurons,alex_num,27,13,2,0,3);


    /*** Alex 3rd conv + Res 3rd conv ***/
    // gridSize = 48;
    dim3 Block7(gridSize,1,1);
    dim3 Thread7(32,blockSize,1);

    a_in_fm         = 13;   a_in_channel    = 192;
    a_out_fm        = 13;   a_out_channel   = 384;
    a_ker_fm        = 3;
    a_str           = 1;    a_pad           = 1;    a_relu          = true;

    r_in_fm         = 56;   r_in_channel    = 64;
    r_out_fm        = 56;   r_out_channel   = 64;
    r_ker_fm        = 3;    
    r_str           = 1;    r_pad           = 1;    r_relu          = false;
    
    alexNumOps = (a_out_fm*a_out_fm*a_out_channel - 1) / (gridSize * 32 * alexWarpNum) + 1;
    resNumOps = (r_out_fm*r_out_fm*r_out_channel - 1) / (gridSize * 32 * resWarpNum) + 1;

    fused_two_conv_block<<<Block7, Thread7>>>(Alex_Layer3_bias,Alex_Layer3_Weights,Res_Layer3_Weights,
                                    Alex_Layer3_Neurons,Res_Layer3_Neurons,
                                    Alex_Layer4_Neurons,Res_Layer3_bn,
                                    alex_num,res_num,
                                    a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
                                    r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
                                    alexWarpNum,resWarpNum,
                                    alexNumOps,resNumOps,
                                    false);

    /* Res 3rd bn */
    dim3 Block8(64,8,8);
    dim3 Thread8(7,7);
 	batchnorm_jjb<<<Block8,Thread8>>>(Res_Layer3_bn,Res_Layer3_basic,res_num,Res_mean3,Res_var3,Res_Layer3_Gamma,Res_Layer3_Beta,56,false);
   
    /* Res 3rd basic */
    dim3 Block9(64,8,8);
    dim3 Thread9(7,7);
    basic_block_jjb<<<Block9,Thread9>>>(Res_Layer2_Neurons,Res_Layer3_basic,Res_Layer4_Neurons,res_num,56,true);


    /*** Alex 4th conv + Res 4th conv ***/
    // gridSize = 48;
    dim3 Block10(gridSize,1,1);
    dim3 Thread10(32,blockSize,1);

    a_in_fm         = 13;   a_in_channel    = 384;
    a_out_fm        = 13;   a_out_channel   = 256;
    a_ker_fm        = 3;
    a_str           = 1;    a_pad           = 1;    a_relu          = true;

    r_in_fm         = 56;   r_in_channel    = 64;
    r_out_fm        = 56;   r_out_channel   = 64;
    r_ker_fm        = 3;    
    r_str           = 1;    r_pad           = 1;    r_relu          = false;
    
    alexNumOps = (a_out_fm*a_out_fm*a_out_channel - 1) / (gridSize * 32 * alexWarpNum) + 1;
    resNumOps = (r_out_fm*r_out_fm*r_out_channel - 1) / (gridSize * 32 * resWarpNum) + 1;

    fused_two_conv_block<<<Block10, Thread10>>>(Alex_Layer4_bias,Alex_Layer4_Weights,Res_Layer4_Weights,
                                    Alex_Layer4_Neurons,Res_Layer4_Neurons,
                                    Alex_Layer5_Neurons,Res_Layer4_bn,
                                    alex_num,res_num,
                                    a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
                                    r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
                                    alexWarpNum,resWarpNum,
                                    alexNumOps,resNumOps,
                                    false);

    /* Res 4th bn */
    dim3 Block11(64,7,7);
    dim3 Thread11(8,8);
	batchnorm_jjb<<<Block11,Thread11>>>(Res_Layer4_bn,Res_Layer5_Neurons,res_num,Res_mean4,Res_var4,Res_Layer4_Gamma,Res_Layer4_Beta,56,true);
	

    /*** Alex 5th conv + Res 5th conv ***/
    // gridSize = 96;
    dim3 Block12(gridSize,1,1);
    dim3 Thread12(32,blockSize,1);

    a_in_fm         = 13;   a_in_channel    = 256;
    a_out_fm        = 13;   a_out_channel   = 256;
    a_ker_fm        = 3;
    a_str           = 1;    a_pad           = 1;    a_relu          = true;

    r_in_fm         = 56;   r_in_channel    = 64;
    r_out_fm        = 56;   r_out_channel   = 64;
    r_ker_fm        = 3;    
    r_str           = 1;    r_pad           = 1;    r_relu          = false;
    
    alexNumOps = (a_out_fm*a_out_fm*a_out_channel - 1) / (gridSize * 32 * alexWarpNum) + 1;
    resNumOps = (r_out_fm*r_out_fm*r_out_channel - 1) / (gridSize * 32 * resWarpNum) + 1;

    fused_two_conv_block<<<Block12, Thread12>>>(Alex_Layer5_bias,Alex_Layer5_Weights,Res_Layer5_Weights,
                                    Alex_Layer5_Neurons,Res_Layer5_Neurons,
                                    Alex_Layer5_pool,Res_Layer5_bn,
                                    alex_num,res_num,
                                    a_in_fm,a_in_channel,a_out_fm,a_out_channel,a_ker_fm,a_str,a_pad,a_relu,
                                    r_in_fm,r_in_channel,r_out_fm,r_out_channel,r_ker_fm,r_str,r_pad,r_relu,
                                    alexWarpNum,resWarpNum,
                                    alexNumOps,resNumOps,
                                    false);
					
    /* Alex 5th max + Res 5th bn */
	dim3 Block13(256,7,7);
	dim3 Thread13(8,8);
	fused_bn_max1<<<Block13,Thread13>>>(Res_Layer5_bn,Alex_Layer5_pool,
	                                    Res_Layer5_basic,Alex_Layer6_Neurons,
	                                    res_num,alex_num,
	                                    Res_mean5,Res_var5,Res_Layer5_Gamma,Res_Layer5_Beta,56,false,
	                                    13,6,2,0,3,
										64,7,8,
	                                    256,1,6);

    /* Res 5th basic */
	dim3 Block14(64,8,8);
    dim3 Thread14(7,7);
    basic_block_jjb<<<Block14,Thread14>>>(Res_Layer4_Neurons,Res_Layer5_basic,Res_Layer6_Neurons,res_num,56,true);

   	/* 6th ~ 10th layer */
    dim3 Block15(128, 4, 4);
    dim3 Thread15(7, 7, 1);

    gridSize = 48;
    blockSize = 32;
    dim3 Block_gms(gridSize, 1, 1);
    dim3 Thread_gms(32, blockSize, 1);
    int numOps = (28*28*128 - 1) / (gridSize * 32 * blockSize) + 1;

    //6th layer
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer6_Neurons,Res_Layer6_Weights,Res_Layer6_bn,res_num,
                                56,64,
                                28,128,
                                3,2,1,false,false,numOps);
    // conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer6_Neurons,Res_Layer6_Weights,Res_Layer6_bn,res_num,56,28,2,1,3,64,false,false);
	batchnorm_jjb<<<Block15,Thread15>>>(Res_Layer6_bn,Res_Layer7_Neurons,res_num,Res_mean6,Res_var6,Res_Layer6_Gamma,Res_Layer6_Beta,28,true);

	//7th layer
	// conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer7_Neurons,Res_Layer7_Weights,Res_Layer7_bn,res_num,28,28,1,1,3,128,false,false);
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer7_Neurons,Res_Layer7_Weights,Res_Layer7_bn,res_num,
                                28,128,
                                28,128,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block15,Thread15>>>(Res_Layer7_bn,Res_Layer7_basic,res_num,Res_mean7,Res_var7,Res_Layer7_Gamma,Res_Layer7_Beta,28,false);

	//Block B output
	// conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer6_Neurons,Res_Block3_Weights,Res_Block3_bn,res_num,56,28,2,0,1,64,false,false); 
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer6_Neurons,Res_Block3_Weights,Res_Block3_bn,res_num,
                                56,64,
                                28,128,
                                1,2,0,false,false,numOps);
	batchnorm_jjb<<<Block15,Thread15>>>(Res_Block3_bn,Res_Block3_basic,res_num,Res_Block3_mean,Res_Block3_var,Res_Block3_Gamma,Res_Block3_Beta,28,false);

	basic_block_jjb<<<Block15,Thread15>>>(Res_Layer7_basic,Res_Block3_basic,Res_Layer8_Neurons,res_num,28,true);

	//8th layer
	// conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer8_Neurons,Res_Layer8_Weights,Res_Layer8_bn,res_num,28,28,1,1,3,128,false,false);
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer8_Neurons,Res_Layer8_Weights,Res_Layer8_bn,res_num,
                                28,128,
                                28,128,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block15,Thread15>>>(Res_Layer8_bn,Res_Layer9_Neurons,res_num,Res_mean8,Res_var8,Res_Layer8_Gamma,Res_Layer8_Beta,28,true);

	//9th layer
	// conv_jjb<<<Block15,Thread15>>>(NULL,Res_Layer9_Neurons,Res_Layer9_Weights,Res_Layer9_bn,res_num,28,28,1,1,3,128,false,false);
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer9_Neurons,Res_Layer9_Weights,Res_Layer9_bn,res_num,
                                28,128,
                                28,128,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block15,Thread15>>>(Res_Layer9_bn,Res_Layer9_basic,res_num,Res_mean9,Res_var9,Res_Layer9_Gamma,Res_Layer9_Beta,28,false);

	basic_block_jjb<<<Block15,Thread15>>>(Res_Layer8_Neurons,Res_Layer9_basic,Res_Layer10_Neurons,res_num,28,true);

    /* 10th ~ 13th  */
    dim3 Block27(256,2,2);
    dim3 Thread27(7,7);

    gridSize = 48;
    blockSize = 16;
    dim3 Block_gms2(gridSize, 1, 1);
    dim3 Thread_gms2(32, blockSize, 1);
    numOps = (14*14*256 - 1) / (gridSize * 32 * blockSize) + 1;

    /* Res 10th conv */
	// conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer10_Neurons,Res_Layer10_Weights,Res_Layer10_bn,res_num,28,14,2,1,3,128,false,false);
    gms_conv<<<Block_gms2,Thread_gms2>>>(NULL,Res_Layer10_Neurons,Res_Layer10_Weights,Res_Layer10_bn,res_num,
                                28,128,
                                14,256,
                                3,2,1,false,false,numOps);
	batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer10_bn,Res_Layer11_Neurons,res_num,Res_mean10,Res_var10,Res_Layer10_Gamma,Res_Layer10_Beta,14,true);

	//11th layer
	// conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer11_Neurons,Res_Layer11_Weights,Res_Layer11_bn,res_num,14,14,1,1,3,256,false,false);
    gms_conv<<<Block_gms2,Thread_gms2>>>(NULL,Res_Layer11_Neurons,Res_Layer11_Weights,Res_Layer11_bn,res_num,
                                14,256,
                                14,256,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer11_bn,Res_Layer11_basic,res_num,Res_mean11,Res_var11,Res_Layer11_Gamma,Res_Layer11_Beta,14,false);

    /* Res 11th bn */
	batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer11_bn,Res_Layer11_basic,res_num,Res_mean11,Res_var11,Res_Layer11_Gamma,Res_Layer11_Beta,14,false);

    /* Res 11th block conv + bn + basic */
	// conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer10_Neurons,Res_Block4_Weights,Res_Block4_bn,res_num,28,14,2,0,1,128,false,false);
    gms_conv<<<Block_gms2,Thread_gms2>>>(NULL,Res_Layer10_Neurons,Res_Block4_Weights,Res_Block4_bn,res_num,
                                28,128,
                                14,256,
                                1,2,0,false,false,numOps);
    batchnorm_jjb<<<Block27,Thread27>>>(Res_Block4_bn,Res_Block4_basic,res_num,Res_Block4_mean,Res_Block4_var,Res_Block4_Gamma,Res_Block4_Beta,14,false);
	basic_block_jjb<<<Block27,Thread27>>>(Res_Layer11_basic,Res_Block4_basic,Res_Layer12_Neurons,res_num,14,true);

	//12th layer
	// conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer12_Neurons,Res_Layer12_Weights,Res_Layer12_bn,res_num,14,14,1,1,3,256,false,false);
    gms_conv<<<Block_gms2,Thread_gms2>>>(NULL,Res_Layer12_Neurons,Res_Layer12_Weights,Res_Layer12_bn,res_num,
                                14,256,
                                14,256,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer12_bn,Res_Layer13_Neurons,res_num,Res_mean12,Res_var12,Res_Layer12_Gamma,Res_Layer12_Beta,14,true);

	//13th layer
	// conv_jjb<<<Block27,Thread27>>>(NULL,Res_Layer13_Neurons,Res_Layer13_Weights,Res_Layer13_bn,res_num,14,14,1,1,3,256,false,false); 
    gms_conv<<<Block_gms2,Thread_gms2>>>(NULL,Res_Layer13_Neurons,Res_Layer13_Weights,Res_Layer13_bn,res_num,
                                14,256,
                                14,256,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block27,Thread27>>>(Res_Layer13_bn,Res_Layer13_basic,res_num,Res_mean13,Res_var13,Res_Layer13_Gamma,Res_Layer13_Beta,14,false);

	basic_block_jjb<<<Block27,Thread27>>>(Res_Layer12_Neurons,Res_Layer13_basic,Res_Layer14_Neurons,res_num,14,true);

    /* Res 14th ~ 17th + 18th avgpooling*/
    dim3 Block39(512,1,1);
    dim3 Thread39(7,7);

    numOps = (7*7*512 - 1) / (gridSize * 32 * blockSize) + 1;

    // Res 14th 
	// conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer14_Neurons,Res_Layer14_Weights,Res_Layer14_bn,res_num,14,7,2,1,3,256,false,false);
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer14_Neurons,Res_Layer14_Weights,Res_Layer14_bn,res_num,
                                14,256,
                                7,512,
                                3,2,1,false,false,numOps);
    batchnorm_jjb<<<Block39,Thread39>>>(Res_Layer14_bn,Res_Layer15_Neurons,res_num,Res_mean14,Res_var14,Res_Layer14_Gamma,Res_Layer14_Beta,7,true);

    // Res 15th
	// conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer15_Neurons,Res_Layer15_Weights,Res_Layer15_bn,res_num,7,7,1,1,3,512,false,false);
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer15_Neurons,Res_Layer15_Weights,Res_Layer15_bn,res_num,
                                7,512,
                                7,512,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block39,Thread39>>>(Res_Layer15_bn,Res_Layer15_basic,res_num,Res_mean15,Res_var15,Res_Layer15_Gamma,Res_Layer15_Beta,7,false);

	//Block D output
	// conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer14_Neurons,Res_Block5_Weights,Res_Block5_bn,res_num,14,7,2,0,1,256,false,false);
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer14_Neurons,Res_Block5_Weights,Res_Block5_bn,res_num,
                                14,256,
                                7,512,
                                1,2,0,false,false,numOps);
    batchnorm_jjb<<<Block39,Thread39>>>(Res_Block5_bn,Res_Block5_basic,res_num,Res_Block5_mean,Res_Block5_var,Res_Block5_Gamma,Res_Block5_Beta,7,false);
	basic_block_jjb<<<Block39,Thread39>>>(Res_Layer15_basic,Res_Block5_basic,Res_Layer16_Neurons,res_num,7,true);

    // Res 16th
	// conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer16_Neurons,Res_Layer16_Weights,Res_Layer16_bn,res_num,7,7,1,1,3,512,false,false);
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer16_Neurons,Res_Layer16_Weights,Res_Layer16_bn,res_num,
                                7,512,
                                7,512,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block39,Thread39>>>(Res_Layer16_bn,Res_Layer17_Neurons,res_num,Res_mean16,Res_var16,Res_Layer16_Gamma,Res_Layer16_Beta,7,true);
	
    // Res 17th
	// conv_jjb<<<Block39,Thread39>>>(NULL,Res_Layer17_Neurons,Res_Layer17_Weights,Res_Layer17_bn,res_num,7,7,1,1,3,512,false,false); 
    gms_conv<<<Block_gms,Thread_gms>>>(NULL,Res_Layer17_Neurons,Res_Layer17_Weights,Res_Layer17_bn,res_num,
                                7,512,
                                7,512,
                                3,1,1,false,false,numOps);
    batchnorm_jjb<<<Block39,Thread39>>>(Res_Layer17_bn,Res_Layer17_basic,res_num,Res_mean17,Res_var17,Res_Layer17_Gamma,Res_Layer17_Beta,7,false);

	basic_block_jjb<<<Block39,Thread39>>>(Res_Layer16_Neurons,Res_Layer17_basic,Res_Layer18_Neurons,res_num,7,true);

    // Res 18th avgpooling
    dim3 Block40(512,1,1);
    dim3 Thread40(1,1);
	globalavg_jjb<<<Block40,Thread40>>>(Res_Layer18_Neurons,Res_FC_Neurons,res_num,7);

    /* Alex 6th fc */
    dim3 block41(4096,1,1);
    dim3 Thread41(1,1);

	fc_jjb<<<block41,Thread41>>>(Alex_Layer6_bias,Alex_Layer6_Neurons,Alex_Layer6_Weights,Alex_Layer7_Neurons,alex_num,(6*6*256),true);
    
	/* Alex 7th fc */
    dim3 block42(4096,1,1);
    dim3 Thread42(1,1);

	fc_jjb<<<block42,Thread42>>>(Alex_Layer7_bias,Alex_Layer7_Neurons,Alex_Layer7_Weights,Alex_Layer8_Neurons,alex_num,4096,true);

	

    /* Alex 8th fc + Res 18th fc */
    dim3 block43(1000,1,1);
    dim3 Thread43(1,1);
    fused_two_fc1<<<block43,Thread43>>>(Alex_Layer8_bias,Res_FC_bias,Alex_Layer8_Weights,Res_FC_Weights,
                                        Alex_Layer8_Neurons,Res_FC_Neurons,
                                        Alex_Result_Neurons,Res_Result_Neurons,
                                        alex_num,res_num,
                                        4096, false,
		                                512,false);



    for(int j = 0; j < alex_num; j++){
        float *Alex_Result_Neurons_CPU = (float *) malloc ((1000) * sizeof(float));
		hipMemcpy(Alex_Result_Neurons_CPU, Alex_Result_Neurons, (1000) * sizeof(float), hipMemcpyDeviceToHost);

		float max_alex = 0.0;
		int index_alex = 0;
		for(int i = 0; i < 1000; i++){
			if(max_alex < Alex_Result_Neurons_CPU[i]){
				max_alex = Alex_Result_Neurons_CPU[i];	
				index_alex = i;
			}
		}

		int line_count_alex = 0;
        char buffer_alex[1000];
        FILE *list_alex = fopen("imagenet1000_clsidx_to_labels.txt","rt");
        while(fgets(buffer_alex, 1000, list_alex) != NULL){
            line_count_alex++;
            if(line_count_alex == (index_alex+1)){
                printf("%f Alex: %s", max_alex, buffer_alex);
                // if (max_alex != 17.64119338989257812500F)
                // {
                //     printf("\n---Alexnet Result---");
                //     printf("\nClass ID: %d\nClass Name: %sProbability: %.20f\n\n", index_alex, buffer_alex, max_alex);
                //     exit(1);
                // }
                // printf("Alexnet: %d, %s", index_alex, buffer_alex);
                break;
            }
        }
        fclose(list_alex);
		// free(Alex_Result_Neurons_CPU);
    }


	for(int j = 0; j < res_num; j++){
        float *Res_Result_Neurons_CPU = (float *) malloc ((1000) * sizeof(float));
		hipMemcpy(Res_Result_Neurons_CPU, Res_Result_Neurons, (1000) * sizeof(float), hipMemcpyDeviceToHost);

		float max_res = 0.0;
        int index_res = 0; 
        for(int i = 0; i < 1000; i++){
            if(max_res < Res_Result_Neurons_CPU[i]){
                max_res = Res_Result_Neurons_CPU[i];	
                index_res = i;
            }
        }	
        int line_count_res = 0;
        char buffer_res[1000];
        FILE *list_res = fopen("imagenet1000_clsidx_to_labels.txt","rt");
        while(fgets(buffer_res, 1000, list_res) != NULL){
            line_count_res++;
            if(line_count_res == (index_res+1)){
                printf("%f Res: %s", max_res, buffer_res);
                // if (max_res != 10.29121589660644531250F)
                // {
                //     printf("\n---Resnet18 Result---");
                //     printf("\nClass ID: %d\nClass Name: %sProbability: %.20f\n\n", index_res, buffer_res, max_res);
                //     exit(1);
                // }
                // printf("Resnet18: %d, %s", index_res, buffer_res);
                break;
            }
        }
        fclose(list_res);
		// free(Res_Result_Neurons_CPU);
    }
}