
#include <hip/hip_runtime.h>
__global__ void fused_lrm_bn1(float *a_in_nu,float *r_in_nu,
float *a_out_nu,float *r_out_nu,
int alex_num,int res_num,
float alpha,float beta1,int local_size,int a_out_fm,
float *mean,float *var,float *gamma,float *beta,int r_out_fm,bool bn_relu,
int model1_bidx,int model1_bidyz,int model1_tidxy,
int model2_bidx,int model2_bidyz,int model2_tidxy)
{
	int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

	if((num_out < model1_bidx) && (row_out_block < model1_bidyz) && (col_out_block < model1_bidyz) && (row_out_thread < model1_tidxy) && (col_out_thread < model1_tidxy))
	{
		int a_row = ((model1_tidxy*row_out_block)+(row_out_thread));
		int a_col = ((model1_tidxy*col_out_block)+(col_out_thread));
	
		int a_out_position = (a_out_fm*a_out_fm*num_out)
						+ (a_out_fm*a_row)
						+ a_col;
		int a_input_position = (a_out_fm*a_row) + a_col;

		int nStart = 0, nEnd = 0;
		nStart=(num_out-2) > 1 ? (num_out-2) : 1 ;
		nEnd=(num_out+2) < gridDim.x ? (num_out+2) : gridDim.x ;

		float a_sum[5] = {0.0};
		float a_result[5] = {0.0}; 
		for(int i = 0; i < alex_num; i++){
			for(int j = (nStart-1); j < (nEnd-1); j++){
				a_sum[i] += pow((a_in_nu[j*a_out_fm*a_out_fm + a_input_position]),2);
			}
			a_result[i] = (a_in_nu[a_out_position]) / (pow(1 + ((alpha/local_size) * a_sum[i]),beta1));
			a_out_nu[a_out_position] = a_result[i];
		}
	}
	if((num_out < model2_bidx) && (row_out_block < model2_bidyz) && (col_out_block < model2_bidyz) && (row_out_thread < model2_tidxy) && (col_out_thread < model2_tidxy))
	{
		int r_row = ((model2_tidxy*(row_out_block))+(row_out_thread));
		int r_col = ((model2_tidxy*(col_out_block))+(col_out_thread));
	
		int r_out_position = (r_out_fm*r_out_fm*num_out)
						+ (r_out_fm*r_row)
						+ r_col;

		float product[5] = {0.0};
		for(int i = 0; i < res_num; i++){
			product[i] = ((r_in_nu[r_out_position] - mean[num_out])/(sqrt(var[num_out] + 1e-5)))*gamma[num_out] + beta[num_out];
			// relu
			if(bn_relu == true){
				if(product[i] < 0)
					product[i] = 0;
			}
			r_out_nu[r_out_position] = product[i];
		}
	}
}

__global__ void fused_max1(float *in_nu1,float *in_nu2,
float *out_nu1,float *out_nu2,
int model_num1,int model_num2,
int in_fm1,int out_fm1,int str1,int pad1,int ker1,
int in_fm2,int out_fm2,int str2,int pad2,int ker2,
int model1_bidx,int model1_bidyz,int model1_tidxy,
int model2_bidx,int model2_bidyz,int model2_tidxy)
{
	int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

	if((blockIdx.x < model1_bidx) && (blockIdx.y < model1_bidyz) && (blockIdx.z < model1_bidyz) && (threadIdx.x < model1_tidxy) && (threadIdx.y < model1_tidxy))
	{
		int row1 = ((model1_tidxy*row_out_block)+(row_out_thread));
		int col1 = ((model1_tidxy*col_out_block)+(col_out_thread));
	
		int out_position1 = (out_fm1*out_fm1*num_out)
						+ (out_fm1*row1)
						+ col1;
		//Stride
		int x_str1 = 0, y_str1 = 0;
		x_str1 = (row1*str1-pad1)*in_fm1;
		x_str1 = x_str1 < 0 ? 0 : x_str1;
		y_str1 = col1*str1-pad1;
		y_str1 = y_str1 < 0 ? 0 : y_str1;

		//Padding
		int loopr1 = ker1, loopc1 = ker1;

		//Upper
		if(row1 < pad1){
			loopr1 = ker1 - pad1;
		}
		//Bottom
		if(row1 >= out_fm1 - pad1){
			loopr1 = in_fm1 - x_str1/in_fm1;
		}
		//Left
		if(col1 < pad1){
			loopc1 = ker1 - pad1;
		}
		//Right
		if(col1 >= out_fm1 - pad1){
			loopc1 = in_fm1 -  y_str1;
		}

		float max1[5] = {0.0};
		for(int i = 0; i < model_num1; i++){
			for(int j = 0; j < loopr1; j++){
				for(int k = 0; k < loopc1; k++){
					if(max1[i] < (in_nu1[num_out*in_fm1*in_fm1 + j*in_fm1 + k + x_str1 + y_str1]))
						max1[i] = in_nu1[num_out*in_fm1*in_fm1 + j*in_fm1 + k + x_str1 + y_str1];
				}
			}
			out_nu1[out_position1] = max1[i];	
		}
	}
	if((blockIdx.x < model2_bidx) && (blockIdx.y < model2_bidyz) && (blockIdx.z < model2_bidyz) && (threadIdx.x < model2_tidxy) && (threadIdx.y < model2_tidxy))
	{
		int row2 = ((model2_tidxy*row_out_block)+(row_out_thread));
		int col2 = ((model2_tidxy*col_out_block)+(col_out_thread));
	
		int out_position2 = (out_fm2*out_fm2*num_out)
						+ (out_fm2*row2)
						+ col2;
		//Stride
		int x_str2 = 0, y_str2 = 0;
		x_str2 = (row2*str2-pad2)*in_fm2;
		x_str2 = x_str2 < 0 ? 0 : x_str2;
		y_str2 = col2*str2-pad2;
		y_str2 = y_str2 < 0 ? 0 : y_str2;

		//Padding
		int loopr2 = ker2, loopc2 = ker2;

		//Upper
		if(row2 < pad2){
			loopr2 = ker2 - pad2;
		}
		//Bottom
		if(row2 >= out_fm2 - pad2){
			loopr2 = in_fm2 - x_str2/in_fm2;
		}
		//Left
		if(col2 < pad2){
			loopc2 = ker2 - pad2;
		}
		//Right
		if(col2 >= out_fm2 - pad2){
			loopc2 = in_fm2 -  y_str2;
		}

		float max2[5] = {0.0};
		for(int i = 0; i < model_num2; i++){
			for(int j = 0; j < loopr2; j++){
				for(int k = 0; k < loopc2; k++){
					if(max2[i] < (in_nu2[num_out*in_fm2*in_fm2 + j*in_fm2 + k + x_str2 + y_str2]))
						max2[i] = in_nu2[num_out*in_fm2*in_fm2 + j*in_fm2 + k + x_str2 + y_str2];
				}
			}
			out_nu2[out_position2] = max2[i];	
		}
	}
}

__global__ void fused_bn_max1(float *in_nu1,float *in_nu2,
float *out_nu1,float *out_nu2,
int model_num1,int model_num2,
float *mean,float *var,float *gamma,float *beta,int out_fm1,bool relu,
int in_fm2,int out_fm2,int str2,int pad2,int ker2,
int model1_bidx,int model1_bidyz,int model1_tidxy,
int model2_bidx,int model2_bidyz,int model2_tidxy)
{
	int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

	if((blockIdx.x < model1_bidx) && (blockIdx.y < model1_bidyz) && (blockIdx.z < model1_bidyz) && (threadIdx.x < model1_tidxy) && (threadIdx.y < model1_tidxy))
	{
		int row1 = ((model1_tidxy*row_out_block)+(row_out_thread));
    	int col1 = ((model1_tidxy*col_out_block)+(col_out_thread));

		int out_position1 = (out_fm1*out_fm1*num_out)
						+ (out_fm1*row1)
						+ col1;

		float product1[5] = {0.0};
		for(int i = 0; i < model_num1; i++){
			product1[i] = ((in_nu1[out_position1] - mean[num_out])/(sqrt(var[num_out] + 1e-5)))*gamma[num_out] + beta[num_out];
			//ReLU
			if(relu == true){
				{
					if(product1[i] < 0)
						product1[i] = 0;
				}
   			}
			out_nu1[out_position1] = product1[i];
		}
	}

	if((blockIdx.x < model2_bidx) && (blockIdx.y < model2_bidyz) && (blockIdx.z < model2_bidyz) && (threadIdx.x < model2_tidxy) && (threadIdx.y < model2_tidxy)){
		

		int row2 = ((model2_tidxy*row_out_block)+(row_out_thread));
    	int col2 = ((model2_tidxy*col_out_block)+(col_out_thread));
	
		int out_position2 = (out_fm2*out_fm2*num_out)
						+ (out_fm2*row2)
						+ col2;
		
		//Stride
		int x_str = 0, y_str = 0;
		x_str = (row2*str2-pad2)*in_fm2;
		x_str = x_str < 0 ? 0 : x_str;
		y_str = col2*str2-pad2;
		y_str = y_str < 0 ? 0 : y_str;

		//Padding
		int loopr = ker2, loopc = ker2;

		//Upper
		if(row2 < pad2){
			loopr = ker2 - pad2;
		}
		//Bottom
		if(row2 >= out_fm2 - pad2){
			loopr = in_fm2 - x_str/in_fm2;
		}
		//Left
		if(col2 < pad2){
			loopc = ker2 - pad2;
		}
		//Right
		if(col2 >= out_fm2 - pad2){
			loopc = in_fm2 -  y_str;
		}

		float max[5] = {0.0};
		for(int i = 0; i < model_num2; i++){
			for(int j = 0; j < loopr; j++){
				for(int k = 0; k < loopc; k++){
					if(max[i] < (in_nu2[num_out*in_fm2*in_fm2 + j*in_fm2 + k + x_str + y_str]))
						max[i] = in_nu2[num_out*in_fm2*in_fm2 + j*in_fm2 + k + x_str + y_str];
				}				
			}
			out_nu2[out_position2] = max[i];
		}
	}
}

__global__ void fused_two_fc1(float *a_bias,float *r_bias,float *a_in_w,float *r_in_w,
float *a_in_nu,float *r_in_nu,
float *a_out_nu,float *r_out_nu,
int alex_num, int res_num,
int a_input, bool a_relu,
int r_input, bool r_relu)
{
	// Only Alexnet + Vgg16
	int num_out = blockIdx.x;

	int a_weight = num_out * a_input;
	float a_result[5] = {0.0};
	for(int i = 0; i < alex_num; i++){
		for(int j = 0; j < a_input; j++){
			a_result[i] += a_in_nu[j] * a_in_w[a_weight+j];
		}
		a_result[i] += a_bias[num_out];

		//ReLU
		if(a_relu == true){
			if(a_result[i] < 0)
				a_result[i] = 0;
		}

		a_out_nu[num_out] = a_result[i];
	}

	int r_weight = num_out * r_input;
	float r_result[5] = {0.0};
	for(int i = 0; i < res_num; i++){
		for(int j = 0; j < r_input; j++){
			r_result[i] += r_in_nu[j] * r_in_w[r_weight+j];
		}
		r_result[i] += r_bias[num_out];

		//ReLU
		if(r_relu == true){
			if(r_result[i] < 0)
				r_result[i] = 0;
		}

		r_out_nu[num_out] = r_result[i];
	}
}

__global__ void fused_first_layer(float *a_bias,float *a_in_w,float *r_in_w,
float *a_in_nu,float *r_in_nu,
float *a_out_nu,float *r_out_nu,
int alex_num,int res_num,
int a_in_fm,int a_out_fm,int a_str,int a_pad,int a_ker,int a_ker_channel,
int r_in_fm,int r_out_fm,int r_str,int r_pad,int r_ker,int r_ker_channel,
int model1_bidyz,int model1_tidxy,
int model2_bidyz,int model2_tidxy)
{
    int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;
	// (blockIdx.x < model1_bidx) &&  
	if((row_out_block < model1_bidyz) && (col_out_block < model1_bidyz) && (row_out_thread < model1_tidxy) && (col_out_thread < model1_tidxy)){
		//// Alexnet ////
		int a_row = ((model1_tidxy*row_out_block)+(row_out_thread));
    	int a_col = ((model1_tidxy*col_out_block)+(col_out_thread));
	  
	  	int a_out_position = (a_out_fm*a_out_fm*num_out)
						+ (a_out_fm*a_row)
						+ a_col;

		//Stride
		int a_x_str = 0, a_y_str = 0;
		a_x_str = 3*(a_row*a_str-a_pad)*a_in_fm; 	// (a_row*a_str-a_pad)*a_in_fm;
		a_x_str = a_x_str < 0 ? 0 : a_x_str;
		a_y_str = 3*(a_col*a_str-a_pad);			// a_col*a_str-a_pad;
		a_y_str = a_y_str < 0 ? 0 : a_y_str;

		//Padding
		int a_x_pad = 0, a_y_pad = 0;
		int a_loopr = a_ker, a_loopc = a_ker;

		//Upper
		if(a_row*a_str < a_pad){
			a_x_pad = a_pad - a_row*a_str;
			a_loopr = a_ker - a_x_pad;
		}
		//Bottom
		if(a_row >= a_out_fm - a_pad){
			a_loopr = a_in_fm - a_x_str/(3*a_in_fm);
		}
		//Left
		if(a_col*a_str < a_pad){
			a_y_pad = a_pad - a_col*a_str;
			a_loopc = a_ker - a_y_pad;
		}
		//Right
		if(a_col >= a_out_fm - a_pad){
			a_loopc = a_in_fm - a_y_str/3;
		}

		float a_product[5] = {0.0};
		for(int i = 0; i < a_loopr; i++){
			for(int j = 0; j < a_loopc; j++){
				for(int k = 0; k < a_ker_channel; k++){
					for(int l = 0; l < alex_num; l++){
						a_product[l] += a_in_nu[i*a_in_fm*a_ker_channel + j*a_ker_channel + k + a_x_str + a_y_str]
										* a_in_w[num_out*a_ker*a_ker*a_ker_channel + i*a_ker + j + k*a_ker*a_ker + a_x_pad*a_ker + a_y_pad];
					}
				}
			}
		}
		for(int i = 0; i < alex_num; i++){

			if(a_loopc > 0 && a_loopr > 0){			
				a_product[i] += a_bias[num_out];
				//ReLU
				if(a_product[i] < 0)
					a_product[i] = 0;
			}
			a_out_nu[a_out_position] = a_product[i];
		}
	}

	if((model1_bidyz <= row_out_block < model1_bidyz + model2_bidyz) && (model1_bidyz <= col_out_block < model1_bidyz + model2_bidyz) && (row_out_thread < model2_tidxy) && (col_out_thread < model2_tidxy)){
		
		int r_row = ((model2_tidxy*(row_out_block-model1_bidyz))+(row_out_thread));
    	int r_col = ((model2_tidxy*(col_out_block-model1_bidyz))+(col_out_thread));

		int r_out_position = (r_out_fm*r_out_fm*num_out)
						+ (r_out_fm*r_row)
						+ r_col;

		//Stride
		int r_x_str = 0, r_y_str = 0;
		r_x_str = 3*(r_row*r_str-r_pad)*r_in_fm; // (r_row*r_str-r_pad)*r_in_fm;
		r_x_str = r_x_str < 0 ? 0 : r_x_str;
		r_y_str = 3*(r_col*r_str-r_pad);		 // r_col*r_str-r_pad;
		r_y_str = r_y_str < 0 ? 0 : r_y_str;

		//Padding
		int r_x_pad = 0, r_y_pad = 0;
		int r_loopr = r_ker, r_loopc = r_ker;

		//Upper
		if(r_row*r_str < r_pad){
			r_x_pad = r_pad - r_row*r_str;
			r_loopr = r_ker - r_x_pad;
		}
		//Bottom
		if(r_row >= r_out_fm - r_pad){
			r_loopr = r_in_fm - r_x_str/(3*r_in_fm);
		}
		//Left
		if(r_col*r_str < r_pad){
			r_y_pad = r_pad - r_col*r_str;
			r_loopc = r_ker - r_y_pad;
		}
		//Right
		if(r_col >= r_out_fm - r_pad){
			r_loopc = r_in_fm -  r_y_str/3;
		}

		float r_product[5] = {0.0};
		for(int i = 0; i < r_loopr; i++){
			for(int j = 0; j < r_loopc; j++){
				for(int k = 0; k < r_ker_channel; k++){
					for(int l = 0; l < res_num; l++){
						r_product[l] += r_in_nu[i*r_in_fm*r_ker_channel + j*r_ker_channel + k + r_x_str + r_y_str]
										* r_in_w[num_out*r_ker*r_ker*r_ker_channel + i*r_ker + j + k*r_ker*r_ker + r_x_pad*r_ker + r_y_pad];
					}	
				}		
			}
		}
		for(int i = 0; i < res_num; i++){
			if(r_loopc > 0 && r_loopr > 0){
				//ReLU
				if(r_product[i] < 0)
					r_product[i] = 0;
			}
			r_out_nu[r_out_position] = r_product[i];
		}
	}
}

__global__ void fused_two_conv(float *a_bias,float *a_in_w,float *r_in_w,
float *a_in_nu,float *r_in_nu,
float *a_out_nu,float *r_out_nu,
int alex_num,int res_num,
int a_in_fm,int a_out_fm,int a_str,int a_pad,int a_ker,int a_ker_channel,bool a_relu,
int r_in_fm,int r_out_fm,int r_str,int r_pad,int r_ker,int r_ker_channel,bool r_relu,
int model1_bidx,int model1_bidyz,int model1_tidxy,
int model2_bidx,int model2_bidyz,int model2_tidxy)
{
	int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

	if((blockIdx.x < model1_bidx) && (blockIdx.y < model1_bidyz) && (blockIdx.z < model1_bidyz) && (threadIdx.x < model1_tidxy) && (threadIdx.y < model1_tidxy))
	{
		int a_row = ((model1_tidxy*row_out_block)+(row_out_thread));
		int a_col = ((model1_tidxy*col_out_block)+(col_out_thread));
	
		int a_out_position = (a_out_fm*a_out_fm*num_out)
						+ (a_out_fm*a_row)
						+ a_col;

		//Stride
		int a_x_str = 0, a_y_str = 0;
		a_x_str = (a_row*a_str-a_pad)*a_in_fm;
		a_x_str = a_x_str < 0 ? 0 : a_x_str;
		a_y_str = a_col*a_str-a_pad;
		a_y_str = a_y_str < 0 ? 0 : a_y_str;

		//Padding
		int a_x_pad = 0, a_y_pad = 0;
		int a_loopr = a_ker, a_loopc = a_ker;

		//Upper
		if(a_row*a_str < a_pad){
			a_x_pad = a_pad - a_row*a_str;
			a_loopr = a_ker - a_x_pad;
		}
		//Bottom
		if(a_row >= a_out_fm - a_pad){
			a_loopr = a_in_fm - a_x_str/a_in_fm;
		}
		//Left
		if(a_col*a_str < a_pad){
			a_y_pad = a_pad - a_col*a_str;
			a_loopc = a_ker - a_y_pad;
		}
		//Right
		if(a_col >= a_out_fm - a_pad){
			a_loopc = a_in_fm -  a_y_str;
		}
		
		float a_product[5] = {0.0};
		for(int i = 0; i < alex_num; i++){
			for(int j = 0; j < a_ker_channel; j++){
				for(int k = 0; k < a_loopr; k++){
					for(int l = 0; l < a_loopc; l++){
						a_product[i] += a_in_nu[a_in_fm*a_in_fm*j + a_in_fm*k + l + (a_x_str + a_y_str)] 
								* a_in_w[num_out*a_ker*a_ker*a_ker_channel + j*a_ker*a_ker + k*a_ker + l + (a_x_pad*a_ker + a_y_pad)];
					}
				}
			}
			if(a_loopc > 0 && a_loopr > 0){
				a_product[i] += a_bias[num_out];

				//ReLU
				if(a_relu == true){
					if(a_product[i] < 0)
						a_product[i] = 0;
				}
				a_out_nu[a_out_position] = a_product[i];
			}
		}
	}

	if((blockIdx.x < model2_bidx) && (blockIdx.y < model2_bidyz) && (blockIdx.z < model2_bidyz) && (threadIdx.x < model2_tidxy) && (threadIdx.y < model2_tidxy))
	{
		int r_row = ((model2_tidxy*row_out_block)+(row_out_thread));
		int r_col = ((model2_tidxy*col_out_block)+(col_out_thread));

		int r_out_position = (r_out_fm*r_out_fm*num_out)
						+ (r_out_fm*r_row)
						+ r_col;

		//Stride
		int r_x_str = 0, r_y_str = 0;
		r_x_str = (r_row*r_str-r_pad)*r_in_fm;
		r_x_str = r_x_str < 0 ? 0 : r_x_str;
		r_y_str = r_col*r_str-r_pad;
		r_y_str = r_y_str < 0 ? 0 : r_y_str;

		//Padding
		int r_x_pad = 0, r_y_pad = 0;
		int r_loopr = r_ker, r_loopc = r_ker;

		//Upper
		if(r_row*r_str < r_pad){
			r_x_pad = r_pad - r_row*r_str;
			r_loopr = r_ker - r_x_pad;
		}
		//Bottom
		if(r_row >= r_out_fm - r_pad){
			r_loopr = r_in_fm - r_x_str/r_in_fm;
		}
		//Left
		if(r_col*r_str < r_pad){
			r_y_pad = r_pad - r_col*r_str;
			r_loopc = r_ker - r_y_pad;
		}
		//Right
		if(r_col >= r_out_fm - r_pad){
			r_loopc = r_in_fm - r_y_str;
		}

		float r_product[5] = {0.0};
		for(int i = 0; i < res_num; i++){
			for(int j = 0; j < r_ker_channel; j++){
				for(int k = 0; k < r_loopr; k++){
					for(int l = 0; l < r_loopc; l++){
						r_product[i] += r_in_nu[r_in_fm*r_in_fm*j + r_in_fm*k + l + r_x_str + r_y_str] 
								*r_in_w[num_out*r_ker*r_ker*r_ker_channel + j*r_ker*r_ker + k*r_ker + l + r_x_pad*r_ker + r_y_pad];
					}
				}
			}
			if(r_loopc > 0 && r_loopr > 0){
				//ReLU
				if(r_relu == true){
					if(r_product[i] < 0)
						r_product[i] = 0;
				}
				r_out_nu[r_out_position] = r_product[i];
			}
		}
	}
}