
#include <hip/hip_runtime.h>
__global__ void first_jjb(float *bias,float *in_nu,float *in_w,float *out_nu,int model_num,
int in_fm,int out_fm,int str,int pad,int ker,int ker_channel,bool b,bool relu)
{
	int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

    int row = ((blockDim.x*row_out_block)+(row_out_thread));
    int col = ((blockDim.y*col_out_block)+(col_out_thread));  

	int out_position = (out_fm*out_fm*num_out)
					 + (out_fm*row)
					 + col;
	//Stride
    int x_str = 0, y_str = 0;
    x_str = 3*(row*str-pad)*in_fm;
    x_str = x_str < 0 ? 0 : x_str;
    y_str = 3*(col*str-pad);
    y_str = y_str < 0 ? 0 : y_str;

	//Padding
	int x_pad = 0, y_pad = 0;
	int loopr = ker, loopc = ker;

	//Upper
	if(row*str < pad){
		x_pad = pad - row*str;
		loopr = ker - x_pad;
	}
	//Bottom
	if(row >= out_fm - pad){
		loopr = in_fm - x_str/(3*in_fm);
	}
	//Left
	if(col*str < pad){
		y_pad = pad - col*str;
		loopc = ker - y_pad;
	}
	//Right
	if(col >= out_fm - pad){
		loopc = in_fm -  y_str/3;
	}

	float product[5] = {0.0};
	for(int l = 0; l < model_num; l++){
		for(int i = 0; i < loopr; i++){
			for(int j = 0; j < loopc; j++){
				for(int k = 0; k < ker_channel; k++){
					product[l] += in_nu[i*in_fm*ker_channel + j*ker_channel + k + x_str + y_str] 
							*in_w[num_out*ker*ker*ker_channel + i*ker + j + k*ker*ker + x_pad*ker + y_pad];
				}
			}
		}
		if(loopc > 0 && loopr > 0){
			if(b == true)
				product[l] += bias[num_out];

			//ReLU
			if(relu == true){
				if(product[l] < 0)
					product[l] = 0;
			}
			out_nu[out_position] = product[l];
		}
	}
}

/* Convolution */
__global__ void conv_jjb(float *bias,float *in_nu,float *in_w,float *out_nu,int model_num,
int in_fm,int out_fm,int str,int pad,int ker,int ker_channel,bool b,bool relu)
{
	int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

    int row = ((blockDim.x*row_out_block)+(row_out_thread));
    int col = ((blockDim.y*col_out_block)+(col_out_thread));  

	int out_position = (out_fm*out_fm*num_out) + (out_fm*row) + col;

    //Stride
    int x_str = 0, y_str = 0;
    x_str = (row*str-pad)*in_fm;
    x_str = x_str < 0 ? 0 : x_str;
    y_str = col*str-pad;
    y_str = y_str < 0 ? 0 : y_str;

	//Padding
	int x_pad = 0, y_pad = 0;
	int loopr = ker, loopc = ker;

	//Upper
	if(row*str < pad){
		x_pad = pad - row*str;
		loopr = ker - x_pad;
	}
	//Bottom
	if(row >= out_fm - pad){
		loopr = in_fm - x_str/in_fm;
	}
	//Left
	if(col*str < pad){
		y_pad = pad - col*str;
		loopc = ker - y_pad;
	}
	//Right
	if(col >= out_fm - pad){
		loopc = in_fm -  y_str;
	}

	float product[5] = {0.0};
	for(int l = 0; l < model_num; l++){
		for(int i = 0; i < ker_channel; i++){
			for(int j = 0; j < loopr; j++){
				for(int k = 0; k < loopc; k++){
					product[l] += in_nu[in_fm*in_fm*i + in_fm*j + k + x_str + y_str] 
							*in_w[num_out*ker_channel*ker*ker + i*ker*ker + j*ker + k + x_pad*ker + y_pad];
				}
			}
		}
		if(loopc > 0 && loopr > 0){
			if(b == true)
				product[l] += bias[num_out];

			//ReLU
			if(relu == true){
				if(product[l] < 0) 
					product[l] = 0;
			}
			out_nu[out_position] = product[l];
		}
	}
}

__global__ void gms_conv(float *bias,float *in_nu,float *in_w,float *out_nu,int model_num,
	int in_fm, int in_channel, 
	int out_fm, int out_channel, 
	int ker, int str, int pad, bool b, bool relu,
	int numOps)
{
	int block_index = blockDim.x * threadIdx.y + threadIdx.x; // block 내에서의 index
	int out_index = blockIdx.x * blockDim.x * blockDim.y + block_index;
	int offset = gridDim.x * blockDim.x * blockDim.y;

	int out_i, out_j, out_k; 	// Alexnet의 output에서의 (i, j, k) 위치
									// i: output kernel
									// j: output row
									// k: output column
	
	int num_elements = out_fm*out_fm*out_channel;

	for (int loops = 0; loops < numOps; loops++)
	{
		if (out_index < num_elements)
		{
			// out_index가 alexnet의 output에서의 좌표가 어디인지 (i, j, k)로 계산하기
			out_i = out_index / (out_fm*out_fm);
			out_j = (out_index % (out_fm*out_fm)) / out_fm;
			out_k = (out_index % (out_fm*out_fm)) % out_fm;

			// a_out(i, j, k)를 계산하기 위해 필요한 input data의 범위 찾기
			int in_k_min = out_k * str - pad;
			int in_k_max = in_k_min + ker - 1;

			int in_j_min = out_j * str - pad;
			int in_j_max = in_j_min + ker - 1;

			bool isElement = false;

			float product[5] = {0.0};

			for (int num = 0; num < model_num; num++)
			{
				
				for (int l = 0; l < in_channel; l++)
				{
					for (int j = in_j_min, ker_j = 0; j <= in_j_max; j++, ker_j++)
					{
						for (int k = in_k_min, ker_k = 0; k <= in_k_max; k++, ker_k++)
						{
							if (j >= 0 && k >= 0 && j < in_fm && k < in_fm)
							{
								product[num] += in_nu[(in_fm*in_fm) * l + (in_fm) * j + k]
												* in_w[(in_channel*ker*ker) * out_i
															+ (ker*ker) * l
															+ (ker) * ker_j
															+ ker_k];

								isElement = true;
							}
						}
					}
				}

				if (isElement)
				{
					if (b)
						product[num] += bias[out_i];

					if (relu && product[num] < 0)
						product[num] = 0;

					out_nu[out_index] = product[num];
				}
			}
		}

		out_index += offset;
	}
}

// __global__ void conv_jjb1(float *bias,float *in_nu,float *in_w,float *out_nu,int model_num,
// int in_fm,int out_fm,int str,int pad,int ker,int ker_channel,bool b,bool relu)
// {
// 	int num_out = blockIdx.x+64;
// 	int row_out_block = blockIdx.y;
// 	int col_out_block = blockIdx.z;
// 	int row_out_thread = threadIdx.x;
// 	int col_out_thread = threadIdx.y;

//     int row = ((blockDim.x*row_out_block)+(row_out_thread));
//     int col = ((blockDim.y*col_out_block)+(col_out_thread));  

// 	int out_position = (out_fm*out_fm*num_out) + (out_fm*row) + col;

//     //Stride
//     int x_str = 0, y_str = 0;
//     x_str = (row*str-pad)*in_fm;
//     x_str = x_str < 0 ? 0 : x_str;
//     y_str = col*str-pad;
//     y_str = y_str < 0 ? 0 : y_str;

// 	//Padding
// 	int x_pad = 0, y_pad = 0;
// 	int loopr = ker, loopc = ker;

// 	//Upper
// 	if(row*str < pad){
// 		x_pad = pad - row*str;
// 		loopr = ker - x_pad;
// 	}
// 	//Bottom
// 	if(row >= out_fm - pad){
// 		loopr = in_fm - x_str/in_fm;
// 	}
// 	//Left
// 	if(col*str < pad){
// 		y_pad = pad - col*str;
// 		loopc = ker - y_pad;
// 	}
// 	//Right
// 	if(col >= out_fm - pad){
// 		loopc = in_fm -  y_str;
// 	}

// 	float product[5] = {0.0};
// 	for(int l = 0; l < model_num; l++){
// 		for(int i = 0; i < ker_channel; i++){
// 			for(int j = 0; j < loopr; j++){
// 				for(int k = 0; k < loopc; k++){
// 					product[l] += in_nu[in_fm*in_fm*i + in_fm*j + k + x_str + y_str] 
// 							*in_w[num_out*ker_channel*ker*ker + i*ker*ker + j*ker + k + x_pad*ker + y_pad];
// 				}
// 			}
// 		}
// 		if(loopc > 0 && loopr > 0){
// 			if(b == true)
// 				product[l] += bias[num_out];

// 			//ReLU
// 			if(relu == true){
// 				if(product[l] < 0) 
// 					product[l] = 0;
// 			}
// 			out_nu[out_position] = product[l];
// 		}
// 	}
// }

/* Local Response Normalization */
__global__ void norm_jjb(float *in_nu,float *out_nu,int model_num,
float alpha,float beta,int local_size,int out_fm)
{
	int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

    int row = ((blockDim.x*row_out_block)+(row_out_thread));
    int col = ((blockDim.y*col_out_block)+(col_out_thread));  

	int out_position = (out_fm*out_fm*num_out) + (out_fm*row) + col;

    int input_position = (out_fm*row) + col;

    int nStart = 0, nEnd = 0;
    nStart=(num_out-2) > 1 ? (num_out-2) : 1 ;
    nEnd=(num_out+2) < gridDim.x ? (num_out+2) : gridDim.x ;

    float sum[5] = {0.0};
    float result[5] = {0.0};  
	for(int i = 0; i < model_num; i++){
		for(int j = (nStart-1); j < (nEnd-1); j++){
			sum[i] += pow((in_nu[j*out_fm*out_fm + input_position]),2);
		}
		result[i] = (in_nu[out_position]) / (pow( 1 + ((alpha/local_size) * sum[i]),beta));
		sum[i] = 0.0;
		out_nu[out_position] = result[i];
	}
}

/* Maxpooling */
__global__ void max_jjb(float *in_nu,float *out_nu,int model_num,
int in_fm,int out_fm,int str,int pad,int ker)
{
    int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

    int row = ((blockDim.x*row_out_block)+(row_out_thread));
    int col = ((blockDim.y*col_out_block)+(col_out_thread));  

	int out_position = (out_fm*out_fm*num_out) + (out_fm*row) + col;

    //Stride
    int x_str = 0, y_str = 0;
    x_str = (row*str-pad)*in_fm;
    x_str = x_str < 0 ? 0 : x_str;
    y_str = col*str-pad;
    y_str = y_str < 0 ? 0 : y_str;

	//Padding
	int loopr = ker, loopc = ker;

	//Upper
	if(row < pad){
		loopr = ker - pad;
	}
	//Bottom
	if(row >= out_fm - pad){
		loopr = in_fm - x_str/in_fm;
	}
	//Left
	if(col < pad){
		loopc = ker - pad;
	}
	//Right
	if(col >= out_fm - pad){
		loopc = in_fm -  y_str;
	}

    float max[5] = {0.0};
	for(int i = 0; i < model_num; i++){
		for(int j = 0; j < loopr; j++){
			for(int k = 0; k < loopc; k++){
				if(max[i] < (in_nu[num_out*in_fm*in_fm + j*in_fm + k + x_str + y_str]))
					max[i] = in_nu[num_out*in_fm*in_fm + j*in_fm + k + x_str + y_str];
			}
		}
    	out_nu[out_position] = max[i];
	}
}

/* Batch Normalization */
__global__ void batchnorm_jjb(float *in_nu,float *out_nu,int model_num,
float *mean,float *var,float *gamma,float *beta,int out_fm,bool relu)
{
    int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

    int row = ((blockDim.x*row_out_block)+(row_out_thread));
    int col = ((blockDim.y*col_out_block)+(col_out_thread));  

	int out_position = (out_fm*out_fm*num_out) + (out_fm*row) + col;

	float product[5] = {0.0};
	for(int i = 0; i < model_num; i++){
		product[i] = ((in_nu[out_position] - mean[num_out])/(sqrt(var[num_out] + 1e-5)))*gamma[num_out] + beta[num_out];
		//ReLU
		if(relu == true){
			if(product[i] < 0)
				product[i] = 0;
		}
		out_nu[out_position] = product[i];
	}
}

/* Basic Block(in resnet) */
__global__ void basic_block_jjb(float *in_nu1,float *in_nu2,float *out_nu,int model_num,
int out_fm,bool relu)
{
	int num_out = blockIdx.x;
	int row_out_block = blockIdx.y;
	int col_out_block = blockIdx.z;
	int row_out_thread = threadIdx.x;
	int col_out_thread = threadIdx.y;

    int row = ((blockDim.x*row_out_block)+(row_out_thread));
    int col = ((blockDim.y*col_out_block)+(col_out_thread));

	int out_position = (out_fm*out_fm*num_out) + (out_fm*row) + col;

	float product[5] = {0.0};
	for(int i = 0; i < model_num; i++){
		product[i] = in_nu1[out_position] + in_nu2[out_position];
		//ReLU
		if(relu == true){
			if(product[i] < 0)
				product[i] = 0;
		}

		out_nu[out_position] = product[i];
	}
}

__global__ void globalavg_jjb(float *in_nu, float *out_nu,int model_num,int in_fm)
{
	int num_out = blockIdx.x;

	float sum[5] = {0.0};
	for(int k = 0; k < model_num; k++){
		for(int i = 0; i < in_fm; i++){
			for(int j = 0; j < in_fm; j++){
				sum[k] += in_nu[num_out*in_fm*in_fm + i*in_fm + j];
			}
		}
		out_nu[num_out] = sum[k]/(in_fm*in_fm);
	}
}

/* Fully Connected */
__global__ void fc_jjb(float *bias,float *in_nu,float *in_w,float *out_nu,int model_num,
int input, bool relu)
{
    int num_out = blockIdx.x;
	int weight = num_out * input;

	float result[5] = {0.0};
	for(int i = 0; i < model_num; i++){
		for(int j = 0; j < input; j++){
			result[i] += in_nu[j] * in_w[weight+j];
		}
		result[i] += bias[num_out];
		//ReLU
		if(relu == true){
			if(result[i] < 0)
				result[i] = 0;
		}
		out_nu[num_out] = result[i];
	}
}

