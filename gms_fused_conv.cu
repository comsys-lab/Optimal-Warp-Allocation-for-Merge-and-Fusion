
#include <hip/hip_runtime.h>
__global__ void fused_two_conv_block(float *a_bias,float *a_in_w,float *r_in_w,
	float *a_in_nu,float *r_in_nu,
	float *a_out_nu,float *r_out_nu,
	int alex_num,int res_num,
	int a_in_fm, int a_in_channel, int a_out_fm, int a_out_channel, int a_ker, int a_str, int a_pad, bool a_relu,
	int r_in_fm, int r_in_channel, int r_out_fm, int r_out_channel, int r_ker, int r_str, int r_pad, bool r_relu,
	int alexnet_warp, int resnet_warp,
	int alexNumOps, int resNumOps,
	bool first)
{
	int block_index = blockDim.x * threadIdx.y + threadIdx.x; // block 내에서의 index

	if (block_index < alexnet_warp * 32)
	{
		int out_index = blockIdx.x * (blockDim.x * alexnet_warp) // output array에서의 index (1차원)
						+ block_index;

		int offset = gridDim.x * blockDim.x * alexnet_warp;

		int a_out_i, a_out_j, a_out_k; 	// Alexnet의 output에서의 (i, j, k) 위치
										// i: output kernel
										// j: output row
										// k: output column
		
		int a_num_elements = a_out_fm*a_out_fm*a_out_channel;

		for (int loops = 0; loops < alexNumOps; loops++)
		{
			if (out_index < a_num_elements)
			{
				// out_index가 alexnet의 output에서의 좌표가 어디인지 (i, j, k)로 계산하기
				a_out_i = out_index / (a_out_fm*a_out_fm);
				a_out_j = (out_index % (a_out_fm*a_out_fm)) / a_out_fm;
				a_out_k = (out_index % (a_out_fm*a_out_fm)) % a_out_fm;

				// a_out(i, j, k)를 계산하기 위해 필요한 input data의 범위 찾기
				int a_in_k_min = a_out_k * a_str - a_pad;
				int a_in_k_max = a_in_k_min + a_ker - 1;

				int a_in_j_min = a_out_j * a_str - a_pad;
				int a_in_j_max = a_in_j_min + a_ker - 1;

				bool isElement = false;

				float a_product[5] = {0.0};
				for (int num = 0; num < alex_num; num++)
				{
					for (int l = 0; l < a_in_channel; l++)
					{
						for (int j = a_in_j_min, ker_j = 0; j <= a_in_j_max; j++, ker_j++)
						{
							for (int k = a_in_k_min, ker_k = 0; k <= a_in_k_max; k++, ker_k++)
							{
								if (j >= 0 && k >= 0 && j < a_in_fm && k < a_in_fm)
								{
									int idx;

									if (first)	idx = (a_in_fm*a_in_channel) * j + (a_in_channel) * k + l;
									else		idx = (a_in_fm*a_in_fm) * l + (a_in_fm) * j	+ k;

									a_product[num] += a_in_nu[idx]
													* a_in_w[(a_in_channel*a_ker*a_ker) * a_out_i
																+ (a_ker*a_ker) * l
																+ (a_ker) * ker_j
																+ ker_k];

									isElement = true;
								}
							}
						}
					}
					if (isElement)
					{
						a_product[num] += a_bias[a_out_i];

						if (a_relu && a_product[num] < 0)
							a_product[num] = 0;
						a_out_nu[out_index] = a_product[num];
					}

				}
			}

			out_index += offset;
		}
	}
	else
	{
		int out_index = blockIdx.x * (blockDim.x * resnet_warp) // output array에서의 index (1차원)
						+ block_index - alexnet_warp * 32;

		int offset = gridDim.x * blockDim.x * resnet_warp;

		int r_out_i, r_out_j, r_out_k; 	// Alexnet의 output에서의 (i, j, k) 위치
										// i: output kernel
										// j: output row
										// k: output column
		
		int r_num_elements = r_out_fm*r_out_fm*r_out_channel;

		for (int loops = 0; loops < resNumOps; loops++)
		{
			if (out_index < r_num_elements)
			{
				// out_index가 alexnet의 output에서의 좌표가 어디인지 (i, j, k)로 계산하기
				r_out_i = out_index / (r_out_fm*r_out_fm);
				r_out_j = (out_index % (r_out_fm*r_out_fm)) / r_out_fm;
				r_out_k = (out_index % (r_out_fm*r_out_fm)) % r_out_fm;

				// r_out(i, j, k)를 계산하기 위해 필요한 input data의 범위 찾기
				int r_in_k_min = r_out_k * r_str - r_pad;
				int r_in_k_max = r_in_k_min + r_ker - 1;

				int r_in_j_min = r_out_j * r_str - r_pad;
				int r_in_j_max = r_in_j_min + r_ker - 1;

				bool isElement = false;

				float r_product[5] = {0.0};
				for (int num = 0; num < res_num; num++)
				{
					for (int l = 0; l < r_in_channel; l++)
					{
						for (int j = r_in_j_min, ker_j = 0; j <= r_in_j_max; j++, ker_j++)
						{
							for (int k = r_in_k_min, ker_k = 0; k <= r_in_k_max; k++, ker_k++)
							{
								if (j >= 0 && k >= 0 && j < r_in_fm && k < r_in_fm)
								{
									int idx;

									if (first)	idx = (r_in_fm*r_in_channel) * j + (r_in_channel) * k + l;
									else		idx = (r_in_fm*r_in_fm) * l + (r_in_fm) * j	+ k;

									r_product[num] += r_in_nu[idx]
													* r_in_w[(r_in_channel*r_ker*r_ker) * r_out_i
																+ (r_ker*r_ker) * l
																+ (r_ker) * ker_j
																+ ker_k];
									isElement = true;
								}
							}
						}
					}
					if (isElement)
					{
						if (r_relu && r_product[num] < 0)
							r_product[num] = 0;
						r_out_nu[out_index] = r_product[num];
					}

				}
			}

			out_index += offset;
		}
	}
}
/*
__global__ void fused_two_conv_block_sh(float *a_bias,float *a_in_w,float *r_in_w,
	float *a_in_nu,float *r_in_nu,
	float *a_out_nu,float *r_out_nu,
	int alex_num,int res_num,
	int a_in_fm, int a_in_channel, int a_out_fm, int a_out_channel, int a_ker, int a_str, int a_pad, bool a_relu,
	int r_in_fm, int r_in_channel, int r_out_fm, int r_out_channel, int r_ker, int r_str, int r_pad, bool r_relu,
	int alexnet_warp, int resnet_warp,
	int alexNumOps, int resNumOps,
	int alexNumOps_sh, int resNumOps_sh,
	bool first)
{
	int block_index = blockDim.x * threadIdx.y + threadIdx.x; // block 내에서의 index

	__shared__ float a_in_w_sh[4800];
	// __shared__ float r_in_w_sh[576];

	if (block_index < alexnet_warp * 32)
	{
		int out_index = blockIdx.x * (blockDim.x * alexnet_warp) // output array에서의 index (1차원)
						+ block_index;

		int offset = gridDim.x * blockDim.x * alexnet_warp;

		int a_out_i, a_out_j, a_out_k; 	// Alexnet의 output에서의 (i, j, k) 위치
										// i: output kernel
										// j: output row
										// k: output column
		
		int a_num_elements = a_out_fm*a_out_fm*a_out_channel;

		for (int loops = 0; loops < alexNumOps; loops++)
		{
			int a_ker_i_start = (out_index - block_index) / (a_out_fm*a_out_fm);
			int a_ker_i_end = (out_index - block_index + 32 * alexnet_warp - 1) / (a_out_fm*a_out_fm);

			int block_index_sh = block_index;

			for (int loops2 = 0; loops2 < alexNumOps_sh; loops2++)
			{
				if (block_index_sh < (a_ker_i_end - a_ker_i_start + 1) * a_in_channel * a_ker * a_ker)
				{
					a_in_w_sh[block_index_sh] = a_in_w[a_ker_i_start*a_in_channel*a_ker*a_ker + block_index_sh];
				}
				block_index_sh += 32 * alexnet_warp;
			}

			__syncthreads();

			if (out_index < a_num_elements)
			{
				// out_index가 alexnet의 output에서의 좌표가 어디인지 (i, j, k)로 계산하기
				a_out_i = out_index / (a_out_fm*a_out_fm);
				a_out_j = (out_index % (a_out_fm*a_out_fm)) / a_out_fm;
				a_out_k = (out_index % (a_out_fm*a_out_fm)) % a_out_fm;

				// a_out(i, j, k)를 계산하기 위해 필요한 input data의 범위 찾기
				int a_in_k_min = a_out_k * a_str - a_pad;
				int a_in_k_max = a_in_k_min + a_ker - 1;

				int a_in_j_min = a_out_j * a_str - a_pad;
				int a_in_j_max = a_in_j_min + a_ker - 1;

				bool isElement = false;

				float a_product[5] = {0.0};
				for (int num = 0; num < alex_num; num++)
				{
					for (int l = 0; l < a_in_channel; l++)
					{
						for (int j = a_in_j_min, ker_j = 0; j <= a_in_j_max; j++, ker_j++)
						{
							for (int k = a_in_k_min, ker_k = 0; k <= a_in_k_max; k++, ker_k++)
							{
								if (j >= 0 && k >= 0 && j < a_in_fm && k < a_in_fm)
								{
									int idx;

									if (first)	idx = (a_in_fm*a_in_channel) * j + (a_in_channel) * k + l;
									else		idx = (a_in_fm*a_in_fm) * l + (a_in_fm) * j	+ k;

									a_product[num] += a_in_nu[idx]
														* a_in_w_sh[(a_in_channel*a_ker*a_ker) * (a_out_i - a_ker_i_start)
																+ (a_ker*a_ker) * l
																+ (a_ker) * ker_j
																+ ker_k];

									isElement = true;
								}
							}
						}
					}
					if (isElement)
					{
						a_product[num] += a_bias[a_out_i];

						if (a_relu)
							a_product[num] = max(0., a_product[num]);
						a_out_nu[out_index] = a_product[num];
					}

				}

				__syncthreads();
			}

			out_index += offset;
		}
	}
	else
	{
		// int out_index = blockIdx.x * (blockDim.x * resnet_warp) // output array에서의 index (1차원)
		// 				+ block_index - alexnet_warp * 32;

		// int offset = gridDim.x * blockDim.x * resnet_warp;

		// int r_out_i, r_out_j, r_out_k; 	// Alexnet의 output에서의 (i, j, k) 위치
		// 								// i: output kernel
		// 								// j: output row
		// 								// k: output column
		
		// int r_num_elements = r_out_fm*r_out_fm*r_out_channel;

		// for (int loops = 0; loops < resNumOps; loops++)
		// {
		// 	if (out_index < r_num_elements)
		// 	{
		// 		// out_index가 alexnet의 output에서의 좌표가 어디인지 (i, j, k)로 계산하기
		// 		r_out_i = out_index / (r_out_fm*r_out_fm);
		// 		r_out_j = (out_index % (r_out_fm*r_out_fm)) / r_out_fm;
		// 		r_out_k = (out_index % (r_out_fm*r_out_fm)) % r_out_fm;

		// 		// r_out(i, j, k)를 계산하기 위해 필요한 input data의 범위 찾기
		// 		int r_in_k_min = r_out_k * r_str - r_pad;
		// 		int r_in_k_max = r_in_k_min + r_ker - 1;

		// 		int r_in_j_min = r_out_j * r_str - r_pad;
		// 		int r_in_j_max = r_in_j_min + r_ker - 1;

		// 		bool isElement = false;

		// 		float r_product[5] = {0.0};
		// 		for (int num = 0; num < res_num; num++)
		// 		{
		// 			for (int l = 0; l < r_in_channel; l++)
		// 			{
		// 				for (int j = r_in_j_min, ker_j = 0; j <= r_in_j_max; j++, ker_j++)
		// 				{
		// 					for (int k = r_in_k_min, ker_k = 0; k <= r_in_k_max; k++, ker_k++)
		// 					{
		// 						if (j >= 0 && k >= 0 && j < r_in_fm && k < r_in_fm)
		// 						{
		// 							int idx;

		// 							if (first)	idx = (r_in_fm*r_in_channel) * j + (r_in_channel) * k + l;
		// 							else		idx = (r_in_fm*r_in_fm) * l + (r_in_fm) * j	+ k;

		// 							r_product[num] += r_in_nu[idx]
		// 											* r_in_w[(r_in_channel*r_ker*r_ker) * r_out_i
		// 														+ (r_ker*r_ker) * l
		// 														+ (r_ker) * ker_j
		// 														+ ker_k];
		// 							isElement = true;
		// 						}
		// 					}
		// 				}
		// 			}
		// 			if (isElement)
		// 			{
		// 				if (r_relu && r_product[num] < 0)
		// 					r_product[num] = 0;
		// 				r_out_nu[out_index] = r_product[num];
		// 			}

		// 		}
		// 	}

		// 	out_index += offset;
		// }
	}
}
*/

__global__ void fused_two_conv_thread(float *a_bias,float *a_in_w,float *r_in_w,
	float *a_in_nu,float *r_in_nu,
	float *a_out_nu,float *r_out_nu,
	int alex_num,int res_num,
	int a_in_fm, int a_in_channel, int a_out_fm, int a_out_channel, int a_ker, int a_str, int a_pad, bool a_relu,
	int r_in_fm, int r_in_channel, int r_out_fm, int r_out_channel, int r_ker, int r_str, int r_pad, bool r_relu,
	int alexnet_thread, int resnet_thread,
	int alexNumOps, int resNumOps)
{
	int block_index = blockDim.x * threadIdx.y + threadIdx.x; // block 내에서의 index

	if (block_index < alexnet_thread * 32)
	{
		int out_index = blockIdx.x * (blockDim.x * alexnet_thread) // output array에서의 index (1차원)
						+ block_index;

		int offset = gridDim.x * blockDim.x * alexnet_thread;

		int a_out_i, a_out_j, a_out_k; 	// Alexnet의 output에서의 (i, j, k) 위치
										// i: output kernel
										// j: output row
										// k: output column
		
		int a_num_elements = a_out_fm*a_out_fm*a_out_channel;

		for (int loops = 0; loops < alexNumOps; loops++)
		{
			if (out_index < a_num_elements)
			{
				// out_index가 alexnet의 output에서의 좌표가 어디인지 (i, j, k)로 계산하기
				a_out_i = out_index / (a_out_fm*a_out_fm);
				a_out_j = (out_index % (a_out_fm*a_out_fm)) / a_out_fm;
				a_out_k = (out_index % (a_out_fm*a_out_fm)) % a_out_fm;

				// a_out(i, j, k)를 계산하기 위해 필요한 input data의 범위 찾기
				int a_in_k_min = a_out_k * a_str - a_pad;
				int a_in_k_max = a_in_k_min + a_ker - 1;

				int a_in_j_min = a_out_j * a_str - a_pad;
				int a_in_j_max = a_in_j_min + a_ker - 1;

				bool isElement = false;

				float a_product[5] = {0.0};
				for (int num = 0; num < alex_num; num++)
				{
					for (int l = 0; l < a_in_channel; l++)
					{
						for (int j = a_in_j_min, ker_j = 0; j <= a_in_j_max; j++, ker_j++)
						{
							for (int k = a_in_k_min, ker_k = 0; k <= a_in_k_max; k++, ker_k++)
							{
								if (j >= 0 && k >= 0 && j < a_in_fm && k < a_in_fm)
								{
									a_product[num] += a_in_nu[(a_in_fm*a_in_fm) * l
																+ (a_in_fm) * j
																+ k]
													* a_in_w[(a_in_channel*a_ker*a_ker) * a_out_i
																+ (a_ker*a_ker) * l
																+ (a_ker) * ker_j
																+ ker_k];
									isElement = true;
								}
							}
						}
					}
					if (isElement)
					{
						a_product[num] += a_bias[a_out_i];

						if (a_relu && a_product[num] < 0)
							a_product[num] = 0;
						a_out_nu[out_index] = a_product[num];
					}

				}
			}

			out_index += offset;
		}
	}
	
	if (block_index < resnet_thread * 32)
	{
		int out_index = blockIdx.x * (blockDim.x * resnet_thread) // output array에서의 index (1차원)
						+ block_index;

		int offset = gridDim.x * blockDim.x * resnet_thread;

		int r_out_i, r_out_j, r_out_k; 	// Alexnet의 output에서의 (i, j, k) 위치
										// i: output kernel
										// j: output row
										// k: output column
		
		int r_num_elements = r_out_fm*r_out_fm*r_out_channel;

		for (int loops = 0; loops < resNumOps; loops++)
		{
			if (out_index < r_num_elements)
			{
				// out_index가 alexnet의 output에서의 좌표가 어디인지 (i, j, k)로 계산하기
				r_out_i = out_index / (r_out_fm*r_out_fm);
				r_out_j = (out_index % (r_out_fm*r_out_fm)) / r_out_fm;
				r_out_k = (out_index % (r_out_fm*r_out_fm)) % r_out_fm;

				// r_out(i, j, k)를 계산하기 위해 필요한 input data의 범위 찾기
				int r_in_k_min = r_out_k * r_str - r_pad;
				int r_in_k_max = r_in_k_min + r_ker - 1;

				int r_in_j_min = r_out_j * r_str - r_pad;
				int r_in_j_max = r_in_j_min + r_ker - 1;

				bool isElement = false;

				float r_product[5] = {0.0};
				for (int num = 0; num < res_num; num++)
				{
					for (int l = 0; l < r_in_channel; l++)
					{
						for (int j = r_in_j_min, ker_j = 0; j <= r_in_j_max; j++, ker_j++)
						{
							for (int k = r_in_k_min, ker_k = 0; k <= r_in_k_max; k++, ker_k++)
							{
								if (j >= 0 && k >= 0 && j < r_in_fm && k < r_in_fm)
								{
									r_product[num] += r_in_nu[(r_in_fm*r_in_fm) * l
																+ (r_in_fm) * j
																+ k]
													* r_in_w[(r_in_channel*r_ker*r_ker) * r_out_i
																+ (r_ker*r_ker) * l
																+ (r_ker) * ker_j
																+ ker_k];
									isElement = true;
								}
							}
						}
					}
					if (isElement)
					{
						if (r_relu && r_product[num] < 0)
							r_product[num] = 0;
						r_out_nu[out_index] = r_product[num];
					}

				}
			}

			out_index += offset;
		}
	}
}