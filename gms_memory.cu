
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>

#define INPUT_SIZE 224*224*3

/* Function to Read Alexnet Input Parameters */
void read_parameter(const char *pFileName,float *layer_parameters)
{
	FILE *fp = fopen(pFileName, "rb");
	int count = 0;
	double temp_num;
	//printf(" File FOUND : %s\n",pFileName);
	while(fscanf(fp, "%lf", &temp_num) == 1){
		layer_parameters[count] = temp_num;
		count++;
	}
	//printf("Final Count : %d\n", count);
	fclose(fp);
}

void Alex_Res_host2gpu(float **Alex_Layer1_Neurons,float **Alex_Layer2_Neurons,float **Alex_Layer3_Neurons,float **Alex_Layer4_Neurons,
					float **Alex_Layer5_Neurons,float **Alex_Layer6_Neurons,float **Alex_Layer7_Neurons,float **Alex_Layer8_Neurons,
                    float **Alex_Layer1_bias,float **Alex_Layer2_bias,float **Alex_Layer3_bias,float **Alex_Layer4_bias,
                    float **Alex_Layer5_bias,float **Alex_Layer6_bias,float **Alex_Layer7_bias,float **Alex_Layer8_bias,
                    float **Alex_Layer1_Weights,float **Alex_Layer2_Weights,float **Alex_Layer3_Weights,float **Alex_Layer4_Weights,
                    float **Alex_Layer5_Weights,float **Alex_Layer6_Weights,float **Alex_Layer7_Weights,float **Alex_Layer8_Weights,
                    float **Alex_Layer1_pool,float **Alex_Layer2_pool,float **Alex_Layer5_pool,
					float **Alex_Layer1_norm,float **Alex_Layer2_norm,float **Alex_Result_Neurons,
					float **Res_Layer1_Neurons,float **Res_Layer2_Neurons,float **Res_Layer3_Neurons,float **Res_Layer4_Neurons,
					float **Res_Layer5_Neurons,float **Res_Layer6_Neurons,float **Res_Layer7_Neurons,float **Res_Layer8_Neurons,
					float **Res_Layer9_Neurons,float **Res_Layer10_Neurons,float **Res_Layer11_Neurons,float **Res_Layer12_Neurons,
					float **Res_Layer13_Neurons,float **Res_Layer14_Neurons,float **Res_Layer15_Neurons,float **Res_Layer16_Neurons,
					float **Res_Layer17_Neurons,float **Res_Layer18_Neurons,
                    float **Res_Layer1_Weights,float **Res_Layer2_Weights,float **Res_Layer3_Weights,float **Res_Layer4_Weights,
                    float **Res_Layer5_Weights,float **Res_Layer6_Weights,float **Res_Layer7_Weights,float **Res_Layer8_Weights,
                    float **Res_Layer9_Weights,float **Res_Layer10_Weights,float **Res_Layer11_Weights,float **Res_Layer12_Weights,
                    float **Res_Layer13_Weights,float **Res_Layer14_Weights,float **Res_Layer15_Weights,float **Res_Layer16_Weights,
                    float **Res_Layer17_Weights,float **Res_Block3_Weights,float **Res_Block4_Weights,float **Res_Block5_Weights,
                    float **Res_Layer1_Gamma,float **Res_Layer2_Gamma,float **Res_Layer3_Gamma,float **Res_Layer4_Gamma,
                    float **Res_Layer5_Gamma,float **Res_Layer6_Gamma,float **Res_Layer7_Gamma,float **Res_Layer8_Gamma,
                    float **Res_Layer9_Gamma,float **Res_Layer10_Gamma,float **Res_Layer11_Gamma,float **Res_Layer12_Gamma,
                    float **Res_Layer13_Gamma,float **Res_Layer14_Gamma,float **Res_Layer15_Gamma,float **Res_Layer16_Gamma,
                    float **Res_Layer17_Gamma,float **Res_Block3_Gamma,float **Res_Block4_Gamma,float **Res_Block5_Gamma,
                    float **Res_Layer1_Beta,float **Res_Layer2_Beta,float**Res_Layer3_Beta,float **Res_Layer4_Beta,
                    float **Res_Layer5_Beta,float **Res_Layer6_Beta,float **Res_Layer7_Beta,float **Res_Layer8_Beta,
                    float **Res_Layer9_Beta,float **Res_Layer10_Beta,float **Res_Layer11_Beta,float **Res_Layer12_Beta,
                    float **Res_Layer13_Beta,float **Res_Layer14_Beta,float **Res_Layer15_Beta,float **Res_Layer16_Beta,
                    float **Res_Layer17_Beta,float **Res_Block3_Beta,float **Res_Block4_Beta,float **Res_Block5_Beta,
                    float **Res_mean1,float **Res_mean2,float **Res_mean3,float **Res_mean4,float **Res_mean5,
                    float **Res_mean6,float **Res_mean7,float **Res_mean8,float **Res_mean9,float **Res_mean10,
                    float **Res_mean11,float **Res_mean12,float **Res_mean13,float **Res_mean14,float **Res_mean15,
                    float **Res_mean16,float **Res_mean17,float **Res_Block3_mean,float **Res_Block4_mean,float **Res_Block5_mean,
                    float **Res_var1,float **Res_var2,float **Res_var3,float **Res_var4,float **Res_var5,
                    float **Res_var6,float **Res_var7,float **Res_var8,float **Res_var9,float **Res_var10,
                    float **Res_var11,float **Res_var12,float **Res_var13,float **Res_var14,float **Res_var15,
                    float **Res_var16,float **Res_var17,float **Res_Block3_var,float **Res_Block4_var,float **Res_Block5_var,
                    float **Res_FC_bias,float **Res_FC_Weights,
					float **Res_Layer3_basic,float **Res_Layer5_basic,float **Res_Layer7_basic,float **Res_Layer9_basic,
					float **Res_Layer11_basic,float **Res_Layer13_basic,float **Res_Layer15_basic,float **Res_Layer17_basic,
					float **Res_Block3_basic,float **Res_Block4_basic,float **Res_Block5_basic,
					float **Res_Layer1_bn,float **Res_Layer2_bn,float **Res_Layer3_bn,float **Res_Layer4_bn,
					float **Res_Layer5_bn,float **Res_Layer6_bn,float **Res_Layer7_bn,float **Res_Layer8_bn,
					float **Res_Layer9_bn,float **Res_Layer10_bn,float **Res_Layer11_bn,float **Res_Layer12_bn,
					float **Res_Layer13_bn,float **Res_Layer14_bn,float **Res_Layer15_bn,float **Res_Layer16_bn,
					float **Res_Layer17_bn,float **Res_Block3_bn,float **Res_Block4_bn,float **Res_Block5_bn,
					float **Res_Layer1_pool,float **Res_FC_Neurons,float **Res_Result_Neurons)
{
	/** Alexnet host2gpu **/
	float *Alex_Layer1_Neurons_CPU = (float*) malloc (INPUT_SIZE * sizeof(float));
	read_parameter("data_alexnet/input_cat1.txt", Alex_Layer1_Neurons_CPU);

	float *Alex_Layer1_bias_CPU = (float*) malloc (64 * sizeof(float));
	float *Alex_Layer2_bias_CPU = (float*) malloc (192 * sizeof(float));
	float *Alex_Layer3_bias_CPU = (float*) malloc (384 * sizeof(float));
	float *Alex_Layer4_bias_CPU = (float*) malloc (256 * sizeof(float));
	float *Alex_Layer5_bias_CPU = (float*) malloc (256 * sizeof(float));
	float *Alex_Layer6_bias_CPU = (float*) malloc (4096 * sizeof(float));
	float *Alex_Layer7_bias_CPU = (float*) malloc (4096 * sizeof(float));
	float *Alex_Layer8_bias_CPU = (float*) malloc (1000 * sizeof(float));

	float *Alex_Layer1_Weights_CPU = (float*) malloc (64*11*11*3 * sizeof(float));
	float *Alex_Layer2_Weights_CPU = (float*) malloc (192*5*5*64 * sizeof(float));
	float *Alex_Layer3_Weights_CPU = (float*) malloc (384*3*3*192 * sizeof(float));
	float *Alex_Layer4_Weights_CPU = (float*) malloc (256*3*3*384 * sizeof(float));
	float *Alex_Layer5_Weights_CPU = (float*) malloc (256*3*3*256 * sizeof(float));
	float *Alex_Layer6_Weights_CPU = (float*) malloc (4096*256*6*6 * sizeof(float));
	float *Alex_Layer7_Weights_CPU = (float*) malloc (4096*4096 * sizeof(float));
	float *Alex_Layer8_Weights_CPU = (float*) malloc (1000*4096 * sizeof(float));

	read_parameter("data_alexnet/bias1.txt", Alex_Layer1_bias_CPU);
	read_parameter("data_alexnet/bias2.txt", Alex_Layer2_bias_CPU);
	read_parameter("data_alexnet/bias3.txt", Alex_Layer3_bias_CPU);
	read_parameter("data_alexnet/bias4.txt", Alex_Layer4_bias_CPU);
	read_parameter("data_alexnet/bias5.txt", Alex_Layer5_bias_CPU);
	read_parameter("data_alexnet/bias6.txt", Alex_Layer6_bias_CPU);
	read_parameter("data_alexnet/bias7.txt", Alex_Layer7_bias_CPU);
	read_parameter("data_alexnet/bias8.txt", Alex_Layer8_bias_CPU);

	read_parameter("data_alexnet/conv1.txt", Alex_Layer1_Weights_CPU);
	read_parameter("data_alexnet/conv2.txt", Alex_Layer2_Weights_CPU);
	read_parameter("data_alexnet/conv3.txt", Alex_Layer3_Weights_CPU);
	read_parameter("data_alexnet/conv4.txt", Alex_Layer4_Weights_CPU);
	read_parameter("data_alexnet/conv5.txt", Alex_Layer5_Weights_CPU);
	read_parameter("data_alexnet/fc6.txt", Alex_Layer6_Weights_CPU);
	read_parameter("data_alexnet/fc7.txt", Alex_Layer7_Weights_CPU);
	read_parameter("data_alexnet/fc8.txt", Alex_Layer8_Weights_CPU);

    float *Alex_Layer1_Neurons_data;
	float *Alex_Layer1_bias_data, *Alex_Layer2_bias_data, *Alex_Layer3_bias_data, *Alex_Layer4_bias_data, 
			*Alex_Layer5_bias_data, *Alex_Layer6_bias_data, *Alex_Layer7_bias_data, *Alex_Layer8_bias_data;
	float *Alex_Layer1_Weights_data, *Alex_Layer2_Weights_data, *Alex_Layer3_Weights_data, *Alex_Layer4_Weights_data,
			*Alex_Layer5_Weights_data, *Alex_Layer6_Weights_data, *Alex_Layer7_Weights_data, *Alex_Layer8_Weights_data;

	hipMalloc((void**) &Alex_Layer1_Neurons_data, INPUT_SIZE * sizeof(float)); //224*224*3
	hipMalloc((void**) &Alex_Layer1_bias_data, 64 * sizeof(float)); //64
	hipMalloc((void**) &Alex_Layer1_Weights_data, (64*11*11*3) * sizeof(float)); //64*11*11*3 = 23232
	hipMalloc((void**) &Alex_Layer2_bias_data, 192 * sizeof(float)); //192
	hipMalloc((void**) &Alex_Layer2_Weights_data, (192*5*5*64) * sizeof(float)); //192*5*5*64 = 307200
	hipMalloc((void**) &Alex_Layer3_bias_data, 384 * sizeof(float)); //384
	hipMalloc((void**) &Alex_Layer3_Weights_data, (384*3*3*192) * sizeof(float)); //384*3*3*192 = 663552
	hipMalloc((void**) &Alex_Layer4_bias_data, 256 * sizeof(float)); //256
	hipMalloc((void**) &Alex_Layer4_Weights_data, (256*3*3*384) * sizeof(float)); //256*3*3*384 = 884736
	hipMalloc((void**) &Alex_Layer5_bias_data, 256 * sizeof(float)); //256
	hipMalloc((void**) &Alex_Layer5_Weights_data, (256*3*3*256) * sizeof(float)); //256*3*3*256 = 442368
	hipMalloc((void**) &Alex_Layer6_bias_data, 4096 * sizeof(float)); //4096
	hipMalloc((void**) &Alex_Layer6_Weights_data, (4096*256*6*6) * sizeof(float)); //4096*256*6*6 = 37748736
	hipMalloc((void**) &Alex_Layer7_bias_data, 4096 * sizeof(float)); //4096
	hipMalloc((void**) &Alex_Layer7_Weights_data, (4096*4096) * sizeof(float)); //4096*4096 = 16777216
	hipMalloc((void**) &Alex_Layer8_bias_data, 1000 * sizeof(float)); //1000
	hipMalloc((void**) &Alex_Layer8_Weights_data, (1000*4096) * sizeof(float)); //1000*4096 = 4096000
	
	hipMemcpy(Alex_Layer1_Neurons_data, Alex_Layer1_Neurons_CPU, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer1_bias_data, Alex_Layer1_bias_CPU, 64 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer1_Weights_data, Alex_Layer1_Weights_CPU, (64*11*11*3) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer2_bias_data, Alex_Layer2_bias_CPU, 192 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer2_Weights_data, Alex_Layer2_Weights_CPU, (192*5*5*64) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer3_bias_data, Alex_Layer3_bias_CPU, 384 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer3_Weights_data, Alex_Layer3_Weights_CPU, (384*3*3*192) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer4_bias_data, Alex_Layer4_bias_CPU, 256 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer4_Weights_data, Alex_Layer4_Weights_CPU, (256*3*3*384) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer5_bias_data, Alex_Layer5_bias_CPU, 256 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer5_Weights_data, Alex_Layer5_Weights_CPU, (256*3*3*256) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer6_bias_data, Alex_Layer6_bias_CPU, 4096 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer6_Weights_data, Alex_Layer6_Weights_CPU, (4096*256*6*6) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer7_bias_data, Alex_Layer7_bias_CPU, 4096 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer7_Weights_data, Alex_Layer7_Weights_CPU, (4096*4096) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer8_bias_data, Alex_Layer8_bias_CPU, 1000 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Alex_Layer8_Weights_data, Alex_Layer8_Weights_CPU, (1000*4096) * sizeof(float), hipMemcpyHostToDevice);

	*Alex_Layer1_Neurons = Alex_Layer1_Neurons_data;

	*Alex_Layer1_bias = Alex_Layer1_bias_data;
	*Alex_Layer2_bias = Alex_Layer2_bias_data;
	*Alex_Layer3_bias = Alex_Layer3_bias_data;
	*Alex_Layer4_bias = Alex_Layer4_bias_data;
	*Alex_Layer5_bias = Alex_Layer5_bias_data;
	*Alex_Layer6_bias = Alex_Layer6_bias_data;
	*Alex_Layer7_bias = Alex_Layer7_bias_data;
	*Alex_Layer8_bias = Alex_Layer8_bias_data;

	*Alex_Layer1_Weights = Alex_Layer1_Weights_data;
	*Alex_Layer2_Weights = Alex_Layer2_Weights_data;
	*Alex_Layer3_Weights = Alex_Layer3_Weights_data;
	*Alex_Layer4_Weights = Alex_Layer4_Weights_data;
	*Alex_Layer5_Weights = Alex_Layer5_Weights_data;
	*Alex_Layer6_Weights = Alex_Layer6_Weights_data;
	*Alex_Layer7_Weights = Alex_Layer7_Weights_data;
	*Alex_Layer8_Weights = Alex_Layer8_Weights_data;

	free(Alex_Layer1_Neurons_CPU);

	free(Alex_Layer1_bias_CPU);
	free(Alex_Layer2_bias_CPU);
	free(Alex_Layer3_bias_CPU);
	free(Alex_Layer4_bias_CPU);
	free(Alex_Layer5_bias_CPU);
	free(Alex_Layer6_bias_CPU);
	free(Alex_Layer7_bias_CPU);
	free(Alex_Layer8_bias_CPU);

    free(Alex_Layer1_Weights_CPU);
    free(Alex_Layer2_Weights_CPU);
    free(Alex_Layer3_Weights_CPU);
    free(Alex_Layer4_Weights_CPU);
    free(Alex_Layer5_Weights_CPU);
    free(Alex_Layer6_Weights_CPU);
    free(Alex_Layer7_Weights_CPU);
    free(Alex_Layer8_Weights_CPU);

	/** Resnet18 host2gpu **/
	float *Res_Layer1_Neurons_CPU = (float*) malloc (INPUT_SIZE * sizeof(float));
	read_parameter("data_resnet18/input_cat.txt", Res_Layer1_Neurons_CPU);

	float *Res_Layer1_Weights_CPU = (float*) malloc ((7*7*3*64) * sizeof(float)); // = 9,408
	float *Res_Layer2_Weights_CPU = (float*) malloc ((3*3*64*64) * sizeof(float)); // = 36,864
	float *Res_Layer3_Weights_CPU = (float*) malloc ((3*3*64*64) * sizeof(float)); // = 36,864
	float *Res_Layer4_Weights_CPU = (float*) malloc ((3*3*64*64) * sizeof(float)); // = 36,864
	float *Res_Layer5_Weights_CPU = (float*) malloc ((3*3*64*64) * sizeof(float)); // = 36,864
	float *Res_Layer6_Weights_CPU = (float*) malloc ((3*3*64*128) * sizeof(float)); // = 73,728
	float *Res_Layer7_Weights_CPU = (float*) malloc ((3*3*128*128) * sizeof(float)); // = 147,456
	float *Res_Layer8_Weights_CPU = (float*) malloc ((3*3*128*128) * sizeof(float)); // = 147,456
    float *Res_Layer9_Weights_CPU = (float*) malloc ((3*3*128*128) * sizeof(float)); // = 147,456
	float *Res_Layer10_Weights_CPU = (float*) malloc ((3*3*128*256) * sizeof(float)); // = 294,912
	float *Res_Layer11_Weights_CPU = (float*) malloc ((3*3*256*256) * sizeof(float)); // = 589,824
	float *Res_Layer12_Weights_CPU = (float*) malloc ((3*3*256*256) * sizeof(float)); // = 589,824
	float *Res_Layer13_Weights_CPU = (float*) malloc ((3*3*256*256) * sizeof(float)); // = 589,824
	float *Res_Layer14_Weights_CPU = (float*) malloc ((3*3*256*512) * sizeof(float)); // = 1,179,648
	float *Res_Layer15_Weights_CPU = (float*) malloc ((3*3*512*512) * sizeof(float)); // = 2,359,296
	float *Res_Layer16_Weights_CPU = (float*) malloc ((3*3*512*512) * sizeof(float)); // = 2,359,296
	float *Res_Layer17_Weights_CPU = (float*) malloc ((3*3*512*512) * sizeof(float)); // = 2,359,296
	float *Res_Block3_Weights_CPU = (float*) malloc ((1*1*64*128) * sizeof(float)); // = 8,192
	float *Res_Block4_Weights_CPU = (float*) malloc ((1*1*128*256) * sizeof(float)); // = 32,768
	float *Res_Block5_Weights_CPU = (float*) malloc ((1*1*256*512) * sizeof(float)); // = 131,072
   
    float *Res_Layer1_Gamma_CPU = (float*) malloc (64 * sizeof(float));
	float *Res_Layer2_Gamma_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer3_Gamma_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer4_Gamma_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer5_Gamma_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer6_Gamma_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Layer7_Gamma_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Layer8_Gamma_CPU = (float*) malloc (128 * sizeof(float)); 
    float *Res_Layer9_Gamma_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Layer10_Gamma_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer11_Gamma_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer12_Gamma_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer13_Gamma_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer14_Gamma_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Layer15_Gamma_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_Layer16_Gamma_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_Layer17_Gamma_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Block3_Gamma_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Block4_Gamma_CPU = (float*) malloc (256 * sizeof(float));
	float *Res_Block5_Gamma_CPU = (float*) malloc (512 * sizeof(float)); 
    
	float *Res_Layer1_Beta_CPU = (float*) malloc (64 * sizeof(float));
	float *Res_Layer2_Beta_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer3_Beta_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer4_Beta_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer5_Beta_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_Layer6_Beta_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Layer7_Beta_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Layer8_Beta_CPU = (float*) malloc (128 * sizeof(float)); 
    float *Res_Layer9_Beta_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_Layer10_Beta_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer11_Beta_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer12_Beta_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer13_Beta_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_Layer14_Beta_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Layer15_Beta_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_Layer16_Beta_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_Layer17_Beta_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Block3_Beta_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Block4_Beta_CPU = (float*) malloc (256 * sizeof(float));
	float *Res_Block5_Beta_CPU = (float*) malloc (512 * sizeof(float));
   
	float *Res_mean1_CPU = (float*) malloc (64 * sizeof(float));
	float *Res_mean2_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_mean3_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_mean4_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_mean5_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_mean6_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_mean7_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_mean8_CPU = (float*) malloc (128 * sizeof(float)); 
    float *Res_mean9_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_mean10_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_mean11_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_mean12_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_mean13_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_mean14_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_mean15_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_mean16_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_mean17_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Block3_mean_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Block4_mean_CPU = (float*) malloc (256 * sizeof(float));
	float *Res_Block5_mean_CPU = (float*) malloc (512 * sizeof(float));
   
	float *Res_var1_CPU = (float*) malloc (64 * sizeof(float));
	float *Res_var2_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_var3_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_var4_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_var5_CPU = (float*) malloc (64 * sizeof(float)); 
	float *Res_var6_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_var7_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_var8_CPU = (float*) malloc (128 * sizeof(float)); 
    float *Res_var9_CPU = (float*) malloc (128 * sizeof(float)); 
	float *Res_var10_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_var11_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_var12_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_var13_CPU = (float*) malloc (256 * sizeof(float)); 
	float *Res_var14_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_var15_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_var16_CPU = (float*) malloc (512 * sizeof(float)); 
	float *Res_var17_CPU = (float*) malloc (512 * sizeof(float));
	float *Res_Block3_var_CPU = (float*) malloc (128 * sizeof(float));
	float *Res_Block4_var_CPU = (float*) malloc (256 * sizeof(float));
	float *Res_Block5_var_CPU = (float*) malloc (512 * sizeof(float));
   
	float *Res_FC_bias_CPU = (float*) malloc (1000* sizeof(float));
	float *Res_FC_Weights_CPU = (float*) malloc ((512*1000) * sizeof(float));

	read_parameter("data_resnet18/conv_data/conv1.txt", Res_Layer1_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv2.txt", Res_Layer2_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv3.txt", Res_Layer3_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv4.txt", Res_Layer4_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv5.txt", Res_Layer5_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv6.txt", Res_Layer6_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv7.txt", Res_Layer7_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv8.txt", Res_Layer8_Weights_CPU);
 	read_parameter("data_resnet18/conv_data/conv9.txt", Res_Layer9_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv10.txt", Res_Layer10_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv11.txt", Res_Layer11_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv12.txt", Res_Layer12_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv13.txt", Res_Layer13_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv14.txt", Res_Layer14_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv15.txt", Res_Layer15_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv16.txt", Res_Layer16_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv17.txt", Res_Layer17_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv_block3.txt", Res_Block3_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv_block4.txt", Res_Block4_Weights_CPU);
	read_parameter("data_resnet18/conv_data/conv_block5.txt", Res_Block5_Weights_CPU);

	read_parameter("data_resnet18/gamma_data/gamma1.txt", Res_Layer1_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma2.txt", Res_Layer2_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma3.txt", Res_Layer3_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma4.txt", Res_Layer4_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma5.txt", Res_Layer5_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma6.txt", Res_Layer6_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma7.txt", Res_Layer7_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma8.txt", Res_Layer8_Gamma_CPU);
 	read_parameter("data_resnet18/gamma_data/gamma9.txt", Res_Layer9_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma10.txt", Res_Layer10_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma11.txt", Res_Layer11_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma12.txt", Res_Layer12_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma13.txt", Res_Layer13_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma14.txt", Res_Layer14_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma15.txt", Res_Layer15_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma16.txt", Res_Layer16_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma17.txt", Res_Layer17_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma_block3.txt", Res_Block3_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma_block4.txt", Res_Block4_Gamma_CPU);
	read_parameter("data_resnet18/gamma_data/gamma_block5.txt", Res_Block5_Gamma_CPU);

	read_parameter("data_resnet18/beta_data/beta1.txt", Res_Layer1_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta2.txt", Res_Layer2_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta3.txt", Res_Layer3_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta4.txt", Res_Layer4_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta5.txt", Res_Layer5_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta6.txt", Res_Layer6_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta7.txt", Res_Layer7_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta8.txt", Res_Layer8_Beta_CPU);
 	read_parameter("data_resnet18/beta_data/beta9.txt", Res_Layer9_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta10.txt", Res_Layer10_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta11.txt", Res_Layer11_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta12.txt", Res_Layer12_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta13.txt", Res_Layer13_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta14.txt", Res_Layer14_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta15.txt", Res_Layer15_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta16.txt", Res_Layer16_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta17.txt", Res_Layer17_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta_block3.txt", Res_Block3_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta_block4.txt", Res_Block4_Beta_CPU);
	read_parameter("data_resnet18/beta_data/beta_block5.txt", Res_Block5_Beta_CPU);

	read_parameter("data_resnet18/mean_data/mean1.txt", Res_mean1_CPU);
	read_parameter("data_resnet18/mean_data/mean2.txt", Res_mean2_CPU);
	read_parameter("data_resnet18/mean_data/mean3.txt", Res_mean3_CPU);
	read_parameter("data_resnet18/mean_data/mean4.txt", Res_mean4_CPU);
	read_parameter("data_resnet18/mean_data/mean5.txt", Res_mean5_CPU);
	read_parameter("data_resnet18/mean_data/mean6.txt", Res_mean6_CPU);
	read_parameter("data_resnet18/mean_data/mean7.txt", Res_mean7_CPU);
	read_parameter("data_resnet18/mean_data/mean8.txt", Res_mean8_CPU);
 	read_parameter("data_resnet18/mean_data/mean9.txt", Res_mean9_CPU);
	read_parameter("data_resnet18/mean_data/mean10.txt", Res_mean10_CPU);
	read_parameter("data_resnet18/mean_data/mean11.txt", Res_mean11_CPU);
	read_parameter("data_resnet18/mean_data/mean12.txt", Res_mean12_CPU);
	read_parameter("data_resnet18/mean_data/mean13.txt", Res_mean13_CPU);
	read_parameter("data_resnet18/mean_data/mean14.txt", Res_mean14_CPU);
	read_parameter("data_resnet18/mean_data/mean15.txt", Res_mean15_CPU);
	read_parameter("data_resnet18/mean_data/mean16.txt", Res_mean16_CPU);
	read_parameter("data_resnet18/mean_data/mean17.txt", Res_mean17_CPU);
	read_parameter("data_resnet18/mean_data/mean_block3.txt", Res_Block3_mean_CPU);
	read_parameter("data_resnet18/mean_data/mean_block4.txt", Res_Block4_mean_CPU);
	read_parameter("data_resnet18/mean_data/mean_block5.txt", Res_Block5_mean_CPU);

	read_parameter("data_resnet18/var_data/var1.txt", Res_var1_CPU);
	read_parameter("data_resnet18/var_data/var2.txt", Res_var2_CPU);
	read_parameter("data_resnet18/var_data/var3.txt", Res_var3_CPU);
	read_parameter("data_resnet18/var_data/var4.txt", Res_var4_CPU);
	read_parameter("data_resnet18/var_data/var5.txt", Res_var5_CPU);
	read_parameter("data_resnet18/var_data/var6.txt", Res_var6_CPU);
	read_parameter("data_resnet18/var_data/var7.txt", Res_var7_CPU);
	read_parameter("data_resnet18/var_data/var8.txt", Res_var8_CPU);
 	read_parameter("data_resnet18/var_data/var9.txt", Res_var9_CPU);
	read_parameter("data_resnet18/var_data/var10.txt", Res_var10_CPU);
	read_parameter("data_resnet18/var_data/var11.txt", Res_var11_CPU);
	read_parameter("data_resnet18/var_data/var12.txt", Res_var12_CPU);
	read_parameter("data_resnet18/var_data/var13.txt", Res_var13_CPU);
	read_parameter("data_resnet18/var_data/var14.txt", Res_var14_CPU);
	read_parameter("data_resnet18/var_data/var15.txt", Res_var15_CPU);
	read_parameter("data_resnet18/var_data/var16.txt", Res_var16_CPU);
	read_parameter("data_resnet18/var_data/var17.txt", Res_var17_CPU);
	read_parameter("data_resnet18/var_data/var_block3.txt", Res_Block3_var_CPU);
	read_parameter("data_resnet18/var_data/var_block4.txt", Res_Block4_var_CPU);
	read_parameter("data_resnet18/var_data/var_block5.txt", Res_Block5_var_CPU);

	read_parameter("data_resnet18/fc_data/fc1_bias.txt", Res_FC_bias_CPU);
	read_parameter("data_resnet18/fc_data/fc1_weight.txt", Res_FC_Weights_CPU);

    float *Res_Layer1_Neurons_data;
	float *Res_Layer1_Weights_data, *Res_Layer2_Weights_data, *Res_Layer3_Weights_data, *Res_Layer4_Weights_data, 
			*Res_Layer5_Weights_data, *Res_Layer6_Weights_data, *Res_Layer7_Weights_data, *Res_Layer8_Weights_data, 
			*Res_Layer9_Weights_data, *Res_Layer10_Weights_data, *Res_Layer11_Weights_data, *Res_Layer12_Weights_data, 
			*Res_Layer13_Weights_data, *Res_Layer14_Weights_data, *Res_Layer15_Weights_data, *Res_Layer16_Weights_data, 
			*Res_Layer17_Weights_data, *Res_Block3_Weights_data, *Res_Block4_Weights_data, *Res_Block5_Weights_data; 
	float *Res_Layer1_Gamma_data, *Res_Layer2_Gamma_data, *Res_Layer3_Gamma_data, *Res_Layer4_Gamma_data,
			*Res_Layer5_Gamma_data, *Res_Layer6_Gamma_data, *Res_Layer7_Gamma_data, *Res_Layer8_Gamma_data,
			*Res_Layer9_Gamma_data, *Res_Layer10_Gamma_data, *Res_Layer11_Gamma_data, *Res_Layer12_Gamma_data,
			*Res_Layer13_Gamma_data, *Res_Layer14_Gamma_data, *Res_Layer15_Gamma_data, *Res_Layer16_Gamma_data,
			*Res_Layer17_Gamma_data, *Res_Block3_Gamma_data, *Res_Block4_Gamma_data, *Res_Block5_Gamma_data;
	float *Res_Layer1_Beta_data, *Res_Layer2_Beta_data, *Res_Layer3_Beta_data, *Res_Layer4_Beta_data,
			*Res_Layer5_Beta_data, *Res_Layer6_Beta_data, *Res_Layer7_Beta_data, *Res_Layer8_Beta_data,
			*Res_Layer9_Beta_data, *Res_Layer10_Beta_data, *Res_Layer11_Beta_data, *Res_Layer12_Beta_data,
			*Res_Layer13_Beta_data, *Res_Layer14_Beta_data, *Res_Layer15_Beta_data, *Res_Layer16_Beta_data,
			*Res_Layer17_Beta_data, *Res_Block3_Beta_data, *Res_Block4_Beta_data, *Res_Block5_Beta_data;
	float *Res_mean1_data, *Res_mean2_data, *Res_mean3_data, *Res_mean4_data, *Res_mean5_data,
			*Res_mean6_data, *Res_mean7_data, *Res_mean8_data, *Res_mean9_data, *Res_mean10_data,
			*Res_mean11_data, *Res_mean12_data, *Res_mean13_data, *Res_mean14_data, *Res_mean15_data,
			*Res_mean16_data, *Res_mean17_data, *Res_Block3_mean_data, *Res_Block4_mean_data, *Res_Block5_mean_data;
	float *Res_var1_data, *Res_var2_data, *Res_var3_data, *Res_var4_data, *Res_var5_data,
			*Res_var6_data, *Res_var7_data, *Res_var8_data, *Res_var9_data, *Res_var10_data,
			*Res_var11_data, *Res_var12_data, *Res_var13_data, *Res_var14_data, *Res_var15_data,
			*Res_var16_data, *Res_var17_data, *Res_Block3_var_data, *Res_Block4_var_data, *Res_Block5_var_data;
	float *Res_FC_bias_data, *Res_FC_Weights_data; 

	hipMalloc((void**) &Res_Layer1_Neurons_data, INPUT_SIZE * sizeof(float)); //224*224*3
	hipMalloc((void**) &Res_Layer1_Weights_data, sizeof(float) * (7*7*3*64));
	hipMalloc((void**) &Res_Layer1_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer1_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean1_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var1_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer2_Weights_data, sizeof(float) * (3*3*64*64));
	hipMalloc((void**) &Res_Layer2_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer2_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean2_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var2_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer3_Weights_data, sizeof(float) * (3*3*64*64));
	hipMalloc((void**) &Res_Layer3_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer3_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean3_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var3_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer4_Weights_data, sizeof(float) * (3*3*64*64));
	hipMalloc((void**) &Res_Layer4_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer4_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean4_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var4_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer5_Weights_data, sizeof(float) * (3*3*64*64));
	hipMalloc((void**) &Res_Layer5_Gamma_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer5_Beta_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_mean5_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_var5_data, sizeof(float) * 64);
	hipMalloc((void**) &Res_Layer6_Weights_data, sizeof(float) * (3*3*64*128));
	hipMalloc((void**) &Res_Layer6_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer6_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_mean6_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_var6_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer7_Weights_data, sizeof(float) * (3*3*128*128));
	hipMalloc((void**) &Res_Layer7_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer7_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_mean7_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_var7_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Block3_Weights_data, sizeof(float) * (1*1*64*128));
	hipMalloc((void**) &Res_Block3_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Block3_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Block3_mean_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Block3_var_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer8_Weights_data, sizeof(float) * (3*3*128*128));
	hipMalloc((void**) &Res_Layer8_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer8_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_mean8_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_var8_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer9_Weights_data, sizeof(float) * (3*3*128*128));
	hipMalloc((void**) &Res_Layer9_Gamma_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer9_Beta_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_mean9_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_var9_data, sizeof(float) * 128);
	hipMalloc((void**) &Res_Layer10_Weights_data, sizeof(float) * (3*3*128*256));
	hipMalloc((void**) &Res_Layer10_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer10_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_mean10_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_var10_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer11_Weights_data, sizeof(float) * (3*3*256*256));	
	hipMalloc((void**) &Res_Layer11_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer11_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_mean11_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_var11_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Block4_Weights_data, sizeof(float) * (1*1*128*256));
	hipMalloc((void**) &Res_Block4_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Block4_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Block4_mean_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Block4_var_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer12_Weights_data, sizeof(float) * (3*3*256*256));
	hipMalloc((void**) &Res_Layer12_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer12_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_mean12_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_var12_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer13_Weights_data, sizeof(float) * (3*3*256*256));
	hipMalloc((void**) &Res_Layer13_Gamma_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer13_Beta_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_mean13_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_var13_data, sizeof(float) * 256);
	hipMalloc((void**) &Res_Layer14_Weights_data, sizeof(float) * (3*3*256*512));
	hipMalloc((void**) &Res_Layer14_Gamma_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer14_Beta_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_mean14_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_var14_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer15_Weights_data, sizeof(float) * (3*3*512*512));
	hipMalloc((void**) &Res_Layer15_Gamma_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer15_Beta_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_mean15_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_var15_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Block5_Weights_data, sizeof(float) * (1*1*256*512));
	hipMalloc((void**) &Res_Block5_Gamma_data, sizeof(float) * 521);
	hipMalloc((void**) &Res_Block5_Beta_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Block5_mean_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Block5_var_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer16_Weights_data, sizeof(float) * (3*3*512*512));
	hipMalloc((void**) &Res_Layer16_Gamma_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer16_Beta_data, sizeof(float) * 512);	
	hipMalloc((void**) &Res_mean16_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_var16_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer17_Weights_data, sizeof(float) * (3*3*512*512));
	hipMalloc((void**) &Res_Layer17_Gamma_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_Layer17_Beta_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_mean17_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_var17_data, sizeof(float) * 512);
	hipMalloc((void**) &Res_FC_bias_data, sizeof(float) * 1000);
	hipMalloc((void**) &Res_FC_Weights_data, sizeof(float) * (512*1000));

	hipMemcpy(Res_Layer1_Neurons_data, Res_Layer1_Neurons_CPU, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer1_Weights_data, Res_Layer1_Weights_CPU, sizeof(float) * (7*7*3*64), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer1_Gamma_data, Res_Layer1_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer1_Beta_data, Res_Layer1_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean1_data, Res_mean1_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var1_data, Res_var1_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer2_Weights_data, Res_Layer2_Weights_CPU, sizeof(float) * (3*3*64*64), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer2_Gamma_data, Res_Layer2_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer2_Beta_data, Res_Layer2_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean2_data, Res_mean2_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var2_data, Res_var2_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer3_Weights_data, Res_Layer3_Weights_CPU, sizeof(float) * (3*3*64*64), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer3_Gamma_data, Res_Layer3_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer3_Beta_data, Res_Layer3_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean3_data, Res_mean3_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var3_data, Res_var3_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer4_Weights_data, Res_Layer4_Weights_CPU, sizeof(float) * (3*3*64*64), hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer4_Gamma_data, Res_Layer4_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer4_Beta_data, Res_Layer4_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean4_data, Res_mean4_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var4_data, Res_var4_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer5_Weights_data, Res_Layer5_Weights_CPU, sizeof(float) * (3*3*64*64), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer5_Gamma_data, Res_Layer5_Gamma_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer5_Beta_data, Res_Layer5_Beta_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean5_data, Res_mean5_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_var5_data, Res_var5_CPU, sizeof(float) * 64, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer6_Weights_data, Res_Layer6_Weights_CPU, sizeof(float) * (3*3*64*128), hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer6_Gamma_data, Res_Layer6_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer6_Beta_data, Res_Layer6_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean6_data, Res_mean6_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_var6_data, Res_var6_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer7_Weights_data, Res_Layer7_Weights_CPU, sizeof(float) * (3*3*128*128), hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer7_Gamma_data, Res_Layer7_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer7_Beta_data, Res_Layer7_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean7_data, Res_mean7_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_var7_data, Res_var7_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_Weights_data, Res_Block3_Weights_CPU, sizeof(float) * (1*1*64*128), hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_Gamma_data, Res_Block3_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_Beta_data, Res_Block3_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_mean_data, Res_Block3_mean_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block3_var_data, Res_Block3_var_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer8_Weights_data, Res_Layer8_Weights_CPU, sizeof(float) * (3*3*128*128), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer8_Gamma_data, Res_Layer8_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer8_Beta_data, Res_Layer8_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);	
	hipMemcpy(Res_mean8_data, Res_mean8_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_var8_data, Res_var8_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer9_Weights_data, Res_Layer9_Weights_CPU, sizeof(float) * (3*3*128*128), hipMemcpyHostToDevice);	
	hipMemcpy(Res_Layer9_Gamma_data, Res_Layer9_Gamma_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer9_Beta_data, Res_Layer9_Beta_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean9_data, Res_mean9_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_var9_data, Res_var9_CPU, sizeof(float) * 128, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer10_Weights_data, Res_Layer10_Weights_CPU, sizeof(float) * (3*3*128*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer10_Gamma_data, Res_Layer10_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer10_Beta_data, Res_Layer10_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean10_data, Res_mean10_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_var10_data, Res_var10_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer11_Weights_data, Res_Layer11_Weights_CPU, sizeof(float) * (3*3*256*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer11_Gamma_data, Res_Layer11_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer11_Beta_data, Res_Layer11_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean11_data, Res_mean11_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_var11_data, Res_var11_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_Weights_data, Res_Block4_Weights_CPU, sizeof(float) * (1*1*128*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_Gamma_data, Res_Block4_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_Beta_data, Res_Block4_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_mean_data, Res_Block4_mean_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block4_var_data, Res_Block4_var_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer12_Weights_data, Res_Layer12_Weights_CPU, sizeof(float) * (3*3*256*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer12_Gamma_data, Res_Layer12_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer12_Beta_data, Res_Layer12_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean12_data, Res_mean12_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_var12_data, Res_var12_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer13_Weights_data, Res_Layer13_Weights_CPU, sizeof(float) * (3*3*256*256), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer13_Gamma_data, Res_Layer13_Gamma_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer13_Beta_data, Res_Layer13_Beta_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean13_data, Res_mean13_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_var13_data, Res_var13_CPU, sizeof(float) * 256, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer14_Weights_data, Res_Layer14_Weights_CPU, sizeof(float) * (3*3*256*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer14_Gamma_data, Res_Layer14_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer14_Beta_data, Res_Layer14_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean14_data, Res_mean14_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_var14_data, Res_var14_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer15_Weights_data, Res_Layer15_Weights_CPU, sizeof(float) * (3*3*512*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer15_Gamma_data, Res_Layer15_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer15_Beta_data, Res_Layer15_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean15_data, Res_mean15_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_var15_data, Res_var15_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_Weights_data, Res_Block5_Weights_CPU, sizeof(float) * (1*1*256*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_Gamma_data, Res_Block5_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_Beta_data, Res_Block5_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_mean_data, Res_Block5_mean_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Block5_var_data, Res_Block5_var_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer16_Weights_data, Res_Layer16_Weights_CPU, sizeof(float) * (3*3*512*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer16_Gamma_data, Res_Layer16_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer16_Beta_data, Res_Layer16_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean16_data, Res_mean16_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_var16_data, Res_var16_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer17_Weights_data, Res_Layer17_Weights_CPU, sizeof(float) * (3*3*512*512), hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer17_Gamma_data, Res_Layer17_Gamma_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_Layer17_Beta_data, Res_Layer17_Beta_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_mean17_data, Res_mean17_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_var17_data, Res_var17_CPU, sizeof(float) * 512, hipMemcpyHostToDevice);
	hipMemcpy(Res_FC_bias_data, Res_FC_bias_CPU, sizeof(float) * 1000, hipMemcpyHostToDevice);
	hipMemcpy(Res_FC_Weights_data, Res_FC_Weights_CPU, sizeof(float) * (512*1000), hipMemcpyHostToDevice);

	*Res_Layer1_Neurons = Res_Layer1_Neurons_data;

	*Res_Layer1_Weights = Res_Layer1_Weights_data;
	*Res_Layer2_Weights = Res_Layer2_Weights_data;
	*Res_Layer3_Weights = Res_Layer3_Weights_data;
	*Res_Layer4_Weights = Res_Layer4_Weights_data;
	*Res_Layer5_Weights = Res_Layer5_Weights_data;
	*Res_Layer6_Weights = Res_Layer6_Weights_data;
	*Res_Layer7_Weights = Res_Layer7_Weights_data;
	*Res_Layer8_Weights = Res_Layer8_Weights_data;
	*Res_Layer9_Weights = Res_Layer9_Weights_data;
	*Res_Layer10_Weights = Res_Layer10_Weights_data;
	*Res_Layer11_Weights = Res_Layer11_Weights_data;
	*Res_Layer12_Weights = Res_Layer12_Weights_data;
	*Res_Layer13_Weights = Res_Layer13_Weights_data;
	*Res_Layer14_Weights = Res_Layer14_Weights_data;
	*Res_Layer15_Weights = Res_Layer15_Weights_data;
	*Res_Layer16_Weights = Res_Layer16_Weights_data;
	*Res_Layer17_Weights = Res_Layer17_Weights_data;
	*Res_Block3_Weights = Res_Block3_Weights_data;
	*Res_Block4_Weights = Res_Block4_Weights_data;
	*Res_Block5_Weights = Res_Block5_Weights_data;
	
	*Res_Layer1_Gamma = Res_Layer1_Gamma_data;
	*Res_Layer2_Gamma = Res_Layer2_Gamma_data;
	*Res_Layer3_Gamma = Res_Layer3_Gamma_data;
	*Res_Layer4_Gamma = Res_Layer4_Gamma_data;
	*Res_Layer5_Gamma = Res_Layer5_Gamma_data;
	*Res_Layer6_Gamma = Res_Layer6_Gamma_data;
	*Res_Layer7_Gamma = Res_Layer7_Gamma_data;
	*Res_Layer8_Gamma = Res_Layer8_Gamma_data;
	*Res_Layer9_Gamma = Res_Layer9_Gamma_data;
	*Res_Layer10_Gamma = Res_Layer10_Gamma_data;
	*Res_Layer11_Gamma = Res_Layer11_Gamma_data;
	*Res_Layer12_Gamma = Res_Layer12_Gamma_data;
	*Res_Layer13_Gamma = Res_Layer13_Gamma_data;
	*Res_Layer14_Gamma = Res_Layer14_Gamma_data;
	*Res_Layer15_Gamma = Res_Layer15_Gamma_data;
	*Res_Layer16_Gamma = Res_Layer16_Gamma_data;
	*Res_Layer17_Gamma = Res_Layer17_Gamma_data;
	*Res_Block3_Gamma = Res_Block3_Gamma_data;
	*Res_Block4_Gamma = Res_Block4_Gamma_data;
	*Res_Block5_Gamma = Res_Block5_Gamma_data;

	*Res_Layer1_Beta = Res_Layer1_Beta_data;
	*Res_Layer2_Beta = Res_Layer2_Beta_data;
	*Res_Layer3_Beta = Res_Layer3_Beta_data;
	*Res_Layer4_Beta = Res_Layer4_Beta_data;
	*Res_Layer5_Beta = Res_Layer5_Beta_data;
	*Res_Layer6_Beta = Res_Layer6_Beta_data;
	*Res_Layer7_Beta = Res_Layer7_Beta_data;
	*Res_Layer8_Beta = Res_Layer8_Beta_data;
	*Res_Layer9_Beta = Res_Layer9_Beta_data;
	*Res_Layer10_Beta = Res_Layer10_Beta_data;
	*Res_Layer11_Beta = Res_Layer11_Beta_data;
	*Res_Layer12_Beta = Res_Layer12_Beta_data;
	*Res_Layer13_Beta = Res_Layer13_Beta_data;
	*Res_Layer14_Beta = Res_Layer14_Beta_data;
	*Res_Layer15_Beta = Res_Layer15_Beta_data;
	*Res_Layer16_Beta = Res_Layer16_Beta_data;
	*Res_Layer17_Beta = Res_Layer17_Beta_data;
	*Res_Block3_Beta = Res_Block3_Beta_data;
	*Res_Block4_Beta = Res_Block4_Beta_data;
	*Res_Block5_Beta = Res_Block5_Beta_data;

	*Res_mean1 = Res_mean1_data;
	*Res_mean2 = Res_mean2_data;
	*Res_mean3 = Res_mean3_data;
	*Res_mean4 = Res_mean4_data;
	*Res_mean5 = Res_mean5_data;
	*Res_mean6 = Res_mean6_data;
	*Res_mean7 = Res_mean7_data;
	*Res_mean8 = Res_mean8_data;
	*Res_mean9 = Res_mean9_data;
	*Res_mean10 = Res_mean10_data;
	*Res_mean11 = Res_mean11_data;
	*Res_mean12 = Res_mean12_data;
	*Res_mean13 = Res_mean13_data;
	*Res_mean14 = Res_mean14_data;
	*Res_mean15 = Res_mean15_data;
	*Res_mean16 = Res_mean16_data;
	*Res_mean17 = Res_mean17_data;
	*Res_Block3_mean = Res_Block3_mean_data;
	*Res_Block4_mean = Res_Block4_mean_data;
	*Res_Block5_mean = Res_Block5_mean_data;

	*Res_var1 = Res_var1_data;
	*Res_var2 = Res_var2_data;
	*Res_var3 = Res_var3_data;
	*Res_var4 = Res_var4_data;
	*Res_var5 = Res_var5_data;
	*Res_var6 = Res_var6_data;
	*Res_var7 = Res_var7_data;
	*Res_var8 = Res_var8_data;
	*Res_var9 = Res_var9_data;
	*Res_var10 = Res_var10_data;
	*Res_var11 = Res_var11_data;
	*Res_var12 = Res_var12_data;
	*Res_var13 = Res_var13_data;
	*Res_var14 = Res_var14_data;
	*Res_var15 = Res_var15_data;
	*Res_var16 = Res_var16_data;
	*Res_var17 = Res_var17_data;
	*Res_Block3_var = Res_Block3_var_data;
	*Res_Block4_var = Res_Block4_var_data;
	*Res_Block5_var = Res_Block5_var_data;

	*Res_FC_bias = Res_FC_bias_data;
	*Res_FC_Weights = Res_FC_Weights_data;

	free(Res_Layer1_Neurons_CPU);

	free(Res_Layer1_Weights_CPU);
    free(Res_Layer2_Weights_CPU);
    free(Res_Layer3_Weights_CPU);
    free(Res_Layer4_Weights_CPU);
    free(Res_Layer5_Weights_CPU);
    free(Res_Layer6_Weights_CPU);
    free(Res_Layer7_Weights_CPU);
    free(Res_Layer8_Weights_CPU);
	free(Res_Layer9_Weights_CPU);
    free(Res_Layer10_Weights_CPU);
    free(Res_Layer11_Weights_CPU);
    free(Res_Layer12_Weights_CPU);
    free(Res_Layer13_Weights_CPU);
    free(Res_Layer14_Weights_CPU);
    free(Res_Layer15_Weights_CPU);
    free(Res_Layer16_Weights_CPU);
	free(Res_Layer17_Weights_CPU);
    free(Res_Block3_Weights_CPU);
    free(Res_Block4_Weights_CPU);
    free(Res_Block5_Weights_CPU);

	free(Res_Layer1_Gamma_CPU);
    free(Res_Layer2_Gamma_CPU);
    free(Res_Layer3_Gamma_CPU);
    free(Res_Layer4_Gamma_CPU);
    free(Res_Layer5_Gamma_CPU);
    free(Res_Layer6_Gamma_CPU);
    free(Res_Layer7_Gamma_CPU);
    free(Res_Layer8_Gamma_CPU);
	free(Res_Layer9_Gamma_CPU);
    free(Res_Layer10_Gamma_CPU);
    free(Res_Layer11_Gamma_CPU);
    free(Res_Layer12_Gamma_CPU);
    free(Res_Layer13_Gamma_CPU);
    free(Res_Layer14_Gamma_CPU);
    free(Res_Layer15_Gamma_CPU);
    free(Res_Layer16_Gamma_CPU);
	free(Res_Layer17_Gamma_CPU);
    free(Res_Block3_Gamma_CPU);
    free(Res_Block4_Gamma_CPU);
    free(Res_Block5_Gamma_CPU);

	free(Res_Layer1_Beta_CPU);
    free(Res_Layer2_Beta_CPU);
    free(Res_Layer3_Beta_CPU);
    free(Res_Layer4_Beta_CPU);
    free(Res_Layer5_Beta_CPU);
    free(Res_Layer6_Beta_CPU);
    free(Res_Layer7_Beta_CPU);
    free(Res_Layer8_Beta_CPU);
	free(Res_Layer9_Beta_CPU);
    free(Res_Layer10_Beta_CPU);
    free(Res_Layer11_Beta_CPU);
    free(Res_Layer12_Beta_CPU);
    free(Res_Layer13_Beta_CPU);
    free(Res_Layer14_Beta_CPU);
    free(Res_Layer15_Beta_CPU);
    free(Res_Layer16_Beta_CPU);
	free(Res_Layer17_Beta_CPU);
    free(Res_Block3_Beta_CPU);
    free(Res_Block4_Beta_CPU);
    free(Res_Block5_Beta_CPU);

	free(Res_mean1_CPU);
	free(Res_mean2_CPU);
	free(Res_mean3_CPU);
	free(Res_mean4_CPU);
	free(Res_mean5_CPU);
	free(Res_mean6_CPU);
	free(Res_mean7_CPU);
	free(Res_mean8_CPU);
	free(Res_mean9_CPU);
	free(Res_mean10_CPU);
	free(Res_mean11_CPU);
	free(Res_mean12_CPU);
	free(Res_mean13_CPU);
	free(Res_mean14_CPU);
	free(Res_mean15_CPU);
	free(Res_mean16_CPU);
	free(Res_mean17_CPU);
	free(Res_Block3_mean_CPU);
	free(Res_Block4_mean_CPU);
	free(Res_Block5_mean_CPU);

	free(Res_var1_CPU);
	free(Res_var2_CPU);
	free(Res_var3_CPU);
	free(Res_var4_CPU);
	free(Res_var5_CPU);
	free(Res_var6_CPU);
	free(Res_var7_CPU);
	free(Res_var8_CPU);
	free(Res_var9_CPU);
	free(Res_var10_CPU);
	free(Res_var11_CPU);
	free(Res_var12_CPU);
	free(Res_var13_CPU);
	free(Res_var14_CPU);
	free(Res_var15_CPU);
	free(Res_var16_CPU);
	free(Res_var17_CPU);
	free(Res_Block3_var_CPU);
	free(Res_Block4_var_CPU);
	free(Res_Block5_var_CPU);

	//추가 cudamalloc
	float *Alex_Layer1_norm_data, *Res_Layer1_bn_data; 
	hipMalloc((void**) &Alex_Layer1_norm_data, (64*55*55) * sizeof(float)); //64*55*55 
	hipMalloc((void**) &Res_Layer1_bn_data, (64*112*112) * sizeof(float)); //64*112*112
	*Alex_Layer1_norm = Alex_Layer1_norm_data;
	*Res_Layer1_bn = Res_Layer1_bn_data;

	float *Alex_Layer1_pool_data, *Res_Layer1_pool_data;
    hipMalloc((void**) &Alex_Layer1_pool_data, (64*55*55) * sizeof(float)); //64*55*55
    hipMalloc((void**) &Res_Layer1_pool_data, (64*112*112) * sizeof(float)); //64*112*112
	*Alex_Layer1_pool = Alex_Layer1_pool_data;
	*Res_Layer1_pool = Res_Layer1_pool_data;

    float *Alex_Layer2_Neurons_data, *Res_Layer2_Neurons_data;
	hipMalloc((void**) &Alex_Layer2_Neurons_data, (64*27*27) * sizeof(float)); //64*27*27
    hipMalloc((void**) &Res_Layer2_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Alex_Layer2_Neurons = Alex_Layer2_Neurons_data;
	*Res_Layer2_Neurons = Res_Layer2_Neurons_data;

    float *Alex_Layer2_norm_data, *Res_Layer2_bn_data;
	hipMalloc((void**) &Alex_Layer2_norm_data, (192*27*27) * sizeof(float)); //192*27*27
    hipMalloc((void**) &Res_Layer2_bn_data, (64*56*56) * sizeof(float)); //64*56*56
	*Alex_Layer2_norm = Alex_Layer2_norm_data;
	*Res_Layer2_bn = Res_Layer2_bn_data;

    float *Alex_Layer2_pool_data, *Res_Layer3_Neurons_data;
    hipMalloc((void**) &Alex_Layer2_pool_data, (192*27*27) * sizeof(float)); //192*27*27
	hipMalloc((void**) &Res_Layer3_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Alex_Layer2_pool = Alex_Layer2_pool_data;
	*Res_Layer3_Neurons = Res_Layer3_Neurons_data;

    float *Alex_Layer3_Neurons_data;
    hipMalloc((void**) &Alex_Layer3_Neurons_data, (192*13*13) * sizeof(float)); //192*13*13
	*Alex_Layer3_Neurons = Alex_Layer3_Neurons_data;

    float *Alex_Layer4_Neurons_data, *Res_Layer3_bn_data;
    hipMalloc((void**) &Alex_Layer4_Neurons_data, (384*13*13) * sizeof(float)); //384*13*13
	hipMalloc((void**) &Res_Layer3_bn_data, (64*56*56) * sizeof(float)); //64*56*56
	*Alex_Layer4_Neurons = Alex_Layer4_Neurons_data;
	*Res_Layer3_bn = Res_Layer3_bn_data;

    float *Res_Layer3_basic_data;
    hipMalloc((void**) &Res_Layer3_basic_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer3_basic = Res_Layer3_basic_data;

    float *Res_Layer4_Neurons_data;
    hipMalloc((void**) &Res_Layer4_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer4_Neurons = Res_Layer4_Neurons_data;

    float *Alex_Layer5_Neurons_data, *Res_Layer4_bn_data;
	hipMalloc((void**) &Alex_Layer5_Neurons_data, (256*13*13) * sizeof(float)); //256*13*13
    hipMalloc((void**) &Res_Layer4_bn_data, (64*56*56) * sizeof(float)); //64*56*56
	*Alex_Layer5_Neurons = Alex_Layer5_Neurons_data;
	*Res_Layer4_bn = Res_Layer4_bn_data;
	
    float *Res_Layer5_Neurons_data;
    hipMalloc((void**) &Res_Layer5_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer5_Neurons = Res_Layer5_Neurons_data;

    float *Alex_Layer5_pool_data, *Res_Layer5_bn_data;
	hipMalloc((void**) &Alex_Layer5_pool_data, (256*13*13) * sizeof(float)); //256*13*13
    hipMalloc((void**) &Res_Layer5_bn_data, (64*56*56) * sizeof(float)); //64*56*56
	*Alex_Layer5_pool = Alex_Layer5_pool_data;
	*Res_Layer5_bn = Res_Layer5_bn_data;

    float *Alex_Layer6_Neurons_data, *Res_Layer5_basic_data;
	hipMalloc((void**) &Alex_Layer6_Neurons_data, (256*6*6) * sizeof(float)); //256*6*6
    hipMalloc((void**) &Res_Layer5_basic_data, (64*56*56) * sizeof(float)); //64*56*56
	*Alex_Layer6_Neurons = Alex_Layer6_Neurons_data;
	*Res_Layer5_basic = Res_Layer5_basic_data;

    float *Res_Layer6_Neurons_data;
    hipMalloc((void**) &Res_Layer6_Neurons_data, (64*56*56) * sizeof(float)); //64*56*56
	*Res_Layer6_Neurons = Res_Layer6_Neurons_data;

    float *Res_Layer6_bn_data;
    hipMalloc((void**) &Res_Layer6_bn_data, sizeof(float) * (128*28*28)); //128*28*28
	*Res_Layer6_bn = Res_Layer6_bn_data;

    float *Res_Layer7_Neurons_data;
    hipMalloc((void**) &Res_Layer7_Neurons_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer7_Neurons = Res_Layer7_Neurons_data;

    float *Res_Layer7_bn_data;
    hipMalloc((void**) &Res_Layer7_bn_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer7_bn = Res_Layer7_bn_data;

    float *Res_Layer7_basic_data;
    hipMalloc((void**) &Res_Layer7_basic_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer7_basic = Res_Layer7_basic_data;

    float *Res_Block3_bn_data, *Res_Block3_basic_data, *Res_Layer8_Neurons_data;
	hipMalloc((void**) &Res_Block3_bn_data, (128*28*28) * sizeof(float)); //128*28*28
	hipMalloc((void**) &Res_Block3_basic_data, (128*28*28) * sizeof(float)); //128*28*28
	hipMalloc((void**) &Res_Layer8_Neurons_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Block3_bn = Res_Block3_bn_data;
	*Res_Block3_basic = Res_Block3_basic_data;
	*Res_Layer8_Neurons = Res_Layer8_Neurons_data;

    float *Res_Layer8_bn_data;
    hipMalloc((void**) &Res_Layer8_bn_data,(128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer8_bn = Res_Layer8_bn_data;

    float *Res_Layer9_Neurons_data;
    hipMalloc((void**) &Res_Layer9_Neurons_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer9_Neurons = Res_Layer9_Neurons_data;

    float *Res_Layer9_bn_data;
    hipMalloc((void**) &Res_Layer9_bn_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer9_bn = Res_Layer9_bn_data;

    float *Res_Layer9_basic_data;
    hipMalloc((void**) &Res_Layer9_basic_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer9_basic = Res_Layer9_basic_data;

    float *Res_Layer10_Neurons_data;
	hipMalloc((void**) &Res_Layer10_Neurons_data, (128*28*28) * sizeof(float)); //128*28*28
	*Res_Layer10_Neurons = Res_Layer10_Neurons_data;

    float *Res_Layer10_bn_data;
    hipMalloc((void**) &Res_Layer10_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer10_bn = Res_Layer10_bn_data;

    float *Res_Layer11_Neurons_data;
    hipMalloc((void**) &Res_Layer11_Neurons_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer11_Neurons = Res_Layer11_Neurons_data;

    float *Res_Layer11_bn_data;
    hipMalloc((void**) &Res_Layer11_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer11_bn = Res_Layer11_bn_data;

    float *Res_Layer11_basic_data;
    hipMalloc((void**) &Res_Layer11_basic_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer11_basic = Res_Layer11_basic_data;

	float *Res_Block4_bn_data, *Res_Block4_basic_data, *Res_Layer12_Neurons_data;
	hipMalloc((void**) &Res_Block4_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	hipMalloc((void**) &Res_Block4_basic_data, (256*14*14) * sizeof(float)); //256*14*14
	hipMalloc((void**) &Res_Layer12_Neurons_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Block4_bn = Res_Block4_bn_data;
	*Res_Block4_basic = Res_Block4_basic_data;
	*Res_Layer12_Neurons = Res_Layer12_Neurons_data;

    float *Res_Layer12_bn_data;
    hipMalloc((void**) &Res_Layer12_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer12_bn = Res_Layer12_bn_data;

    float *Res_Layer13_Neurons_data;
    hipMalloc((void**) &Res_Layer13_Neurons_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer13_Neurons = Res_Layer13_Neurons_data;

    float *Res_Layer13_bn_data;
    hipMalloc((void**) &Res_Layer13_bn_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer13_bn = Res_Layer13_bn_data;

    float *Res_Layer13_basic_data;
    hipMalloc((void**) &Res_Layer13_basic_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer13_basic = Res_Layer13_basic_data;

    float *Res_Layer14_Neurons_data;
    hipMalloc((void**) &Res_Layer14_Neurons_data, (256*14*14) * sizeof(float)); //256*14*14
	*Res_Layer14_Neurons = Res_Layer14_Neurons_data;

    float *Res_Layer14_bn_data, *Res_Layer15_Neurons_data;
	hipMalloc((void**) &Res_Layer14_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer15_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Layer14_bn = Res_Layer14_bn_data;
	*Res_Layer15_Neurons = Res_Layer15_Neurons_data;

    float *Res_Layer15_bn_data, *Res_Layer15_basic_data;
	hipMalloc((void**) &Res_Layer15_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer15_basic_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Layer15_bn = Res_Layer15_bn_data;
	*Res_Layer15_basic = Res_Layer15_basic_data;

	float *Res_Block5_bn_data, *Res_Block5_basic_data, *Res_Layer16_Neurons_data;
	hipMalloc((void**) &Res_Block5_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Block5_basic_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer16_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Block5_bn = Res_Block5_bn_data;
	*Res_Block5_basic = Res_Block5_basic_data;
	*Res_Layer16_Neurons = Res_Layer16_Neurons_data;

	float *Res_Layer16_bn_data, *Res_Layer17_Neurons_data;
	hipMalloc((void**) &Res_Layer16_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer17_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Layer16_bn = Res_Layer16_bn_data;
	*Res_Layer17_Neurons = Res_Layer17_Neurons_data;

    float *Res_Layer17_bn_data, *Res_Layer17_basic_data, *Res_Layer18_Neurons_data;
	hipMalloc((void**) &Res_Layer17_bn_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer17_basic_data, (512*7*7) * sizeof(float)); //512*7*7
	hipMalloc((void**) &Res_Layer18_Neurons_data, (512*7*7) * sizeof(float)); //512*7*7
	*Res_Layer17_bn = Res_Layer17_bn_data;
	*Res_Layer17_basic = Res_Layer17_basic_data;
	*Res_Layer18_Neurons = Res_Layer18_Neurons_data;

    float *Res_FC_Neurons_data;
	hipMalloc((void**) &Res_FC_Neurons_data, 512 * sizeof(float));
	*Res_FC_Neurons = Res_FC_Neurons_data;

    float *Alex_Layer7_Neurons_data;
	hipMalloc((void**) &Alex_Layer7_Neurons_data, 4096 * sizeof(float)); //4096
	*Alex_Layer7_Neurons = Alex_Layer7_Neurons_data;

    float *Alex_Layer8_Neurons_data;
	hipMalloc((void**) &Alex_Layer8_Neurons_data, 4096 * sizeof(float)); //4096
	*Alex_Layer8_Neurons = Alex_Layer8_Neurons_data;

    float *Alex_Result_Neurons_data, *Res_Result_Neurons_data;
	hipMalloc((void**) &Alex_Result_Neurons_data, 1000 * sizeof(float)); //1000
    hipMalloc((void**) &Res_Result_Neurons_data, 1000 * sizeof(float)); //1000
	*Alex_Result_Neurons = Alex_Result_Neurons_data;
	*Res_Result_Neurons = Res_Result_Neurons_data;
}

void Alex_Res_cudafree(float *Alex_Layer1_Neurons,float *Alex_Layer2_Neurons,float *Alex_Layer3_Neurons,float *Alex_Layer4_Neurons,
					float *Alex_Layer5_Neurons,float *Alex_Layer6_Neurons,float *Alex_Layer7_Neurons,float *Alex_Layer8_Neurons,
                    float *Alex_Layer1_bias,float *Alex_Layer2_bias,float *Alex_Layer3_bias,float *Alex_Layer4_bias,
                    float *Alex_Layer5_bias,float *Alex_Layer6_bias,float *Alex_Layer7_bias,float *Alex_Layer8_bias,
                    float *Alex_Layer1_Weights,float *Alex_Layer2_Weights,float *Alex_Layer3_Weights,float *Alex_Layer4_Weights,
                    float *Alex_Layer5_Weights,float * Alex_Layer6_Weights,float *Alex_Layer7_Weights,float *Alex_Layer8_Weights,
                    float *Alex_Layer1_pool,float *Alex_Layer2_pool,float *Alex_Layer5_pool,
					float *Alex_Layer1_norm,float *Alex_Layer2_norm,float *Alex_Result_Neurons,
					float *Res_Layer1_Neurons,float *Res_Layer2_Neurons,float *Res_Layer3_Neurons,float *Res_Layer4_Neurons,
					float *Res_Layer5_Neurons,float *Res_Layer6_Neurons,float *Res_Layer7_Neurons,float *Res_Layer8_Neurons,
					float *Res_Layer9_Neurons,float *Res_Layer10_Neurons,float *Res_Layer11_Neurons,float *Res_Layer12_Neurons,
					float *Res_Layer13_Neurons,float *Res_Layer14_Neurons,float *Res_Layer15_Neurons,float *Res_Layer16_Neurons,
					float *Res_Layer17_Neurons,float *Res_Layer18_Neurons,
                    float *Res_Layer1_Weights,float *Res_Layer2_Weights,float *Res_Layer3_Weights,float *Res_Layer4_Weights,
                    float *Res_Layer5_Weights,float *Res_Layer6_Weights,float *Res_Layer7_Weights,float *Res_Layer8_Weights,
                    float *Res_Layer9_Weights,float *Res_Layer10_Weights,float *Res_Layer11_Weights,float *Res_Layer12_Weights,
                    float *Res_Layer13_Weights,float *Res_Layer14_Weights,float *Res_Layer15_Weights,float *Res_Layer16_Weights,
                    float *Res_Layer17_Weights,float *Res_Block3_Weights,float *Res_Block4_Weights,float *Res_Block5_Weights,
                    float *Res_Layer1_Gamma,float *Res_Layer2_Gamma,float *Res_Layer3_Gamma,float *Res_Layer4_Gamma,
                    float *Res_Layer5_Gamma,float *Res_Layer6_Gamma,float *Res_Layer7_Gamma,float *Res_Layer8_Gamma,
                    float *Res_Layer9_Gamma,float *Res_Layer10_Gamma,float *Res_Layer11_Gamma,float *Res_Layer12_Gamma,
                    float *Res_Layer13_Gamma,float *Res_Layer14_Gamma,float *Res_Layer15_Gamma,float *Res_Layer16_Gamma,
                    float *Res_Layer17_Gamma,float *Res_Block3_Gamma,float *Res_Block4_Gamma,float *Res_Block5_Gamma,
                    float *Res_Layer1_Beta,float *Res_Layer2_Beta,float *Res_Layer3_Beta,float *Res_Layer4_Beta,
                    float *Res_Layer5_Beta,float *Res_Layer6_Beta,float *Res_Layer7_Beta,float *Res_Layer8_Beta,
                    float *Res_Layer9_Beta,float *Res_Layer10_Beta,float *Res_Layer11_Beta,float *Res_Layer12_Beta,
                    float *Res_Layer13_Beta,float *Res_Layer14_Beta,float *Res_Layer15_Beta,float *Res_Layer16_Beta,
                    float *Res_Layer17_Beta,float *Res_Block3_Beta,float *Res_Block4_Beta,float *Res_Block5_Beta,
                    float *Res_mean1,float *Res_mean2,float *Res_mean3,float *Res_mean4,float *Res_mean5,
                    float *Res_mean6,float *Res_mean7,float *Res_mean8,float *Res_mean9,float *Res_mean10,
                    float *Res_mean11,float *Res_mean12,float *Res_mean13,float *Res_mean14,float *Res_mean15,
                    float *Res_mean16,float *Res_mean17,float *Res_Block3_mean,float *Res_Block4_mean,float *Res_Block5_mean,
                    float *Res_var1,float *Res_var2,float *Res_var3,float *Res_var4,float *Res_var5,
                    float *Res_var6,float *Res_var7,float *Res_var8,float *Res_var9,float *Res_var10,
                    float *Res_var11,float *Res_var12,float *Res_var13,float *Res_var14,float *Res_var15,
                    float *Res_var16,float *Res_var17,float *Res_Block3_var,float *Res_Block4_var,float *Res_Block5_var,
                    float *Res_FC_bias,float *Res_FC_Weights,
					float *Res_Layer3_basic,float *Res_Layer5_basic,float *Res_Layer7_basic,float *Res_Layer9_basic,
					float *Res_Layer11_basic,float *Res_Layer13_basic,float *Res_Layer15_basic,float *Res_Layer17_basic,
					float *Res_Block3_basic,float *Res_Block4_basic,float *Res_Block5_basic,
					float *Res_Layer1_bn,float *Res_Layer2_bn,float *Res_Layer3_bn,float *Res_Layer4_bn,
					float *Res_Layer5_bn,float *Res_Layer6_bn,float *Res_Layer7_bn,float *Res_Layer8_bn,
					float *Res_Layer9_bn,float *Res_Layer10_bn,float *Res_Layer11_bn,float *Res_Layer12_bn,
					float *Res_Layer13_bn,float *Res_Layer14_bn,float *Res_Layer15_bn,float *Res_Layer16_bn,
					float *Res_Layer17_bn,float *Res_Block3_bn,float *Res_Block4_bn,float *Res_Block5_bn,
					float *Res_Layer1_pool,float *Res_FC_Neurons,float *Res_Result_Neurons)
{
	hipFree(Alex_Layer1_Neurons);
	hipFree(Alex_Layer2_Neurons);
	hipFree(Alex_Layer3_Neurons);
	hipFree(Alex_Layer4_Neurons);
	hipFree(Alex_Layer5_Neurons);
	hipFree(Alex_Layer6_Neurons);
	hipFree(Alex_Layer7_Neurons);
	hipFree(Alex_Layer8_Neurons);

	hipFree(Alex_Layer1_bias);
	hipFree(Alex_Layer2_bias);
	hipFree(Alex_Layer3_bias);
	hipFree(Alex_Layer4_bias);
	hipFree(Alex_Layer5_bias);
	hipFree(Alex_Layer6_bias);
	hipFree(Alex_Layer7_bias);
	hipFree(Alex_Layer8_bias);

	hipFree(Alex_Layer1_Weights);
	hipFree(Alex_Layer2_Weights);
	hipFree(Alex_Layer3_Weights);
	hipFree(Alex_Layer4_Weights);
	hipFree(Alex_Layer5_Weights);
	hipFree(Alex_Layer6_Weights);
	hipFree(Alex_Layer7_Weights);
	hipFree(Alex_Layer8_Weights);

	hipFree(Alex_Layer1_pool);
	hipFree(Alex_Layer2_pool);
	hipFree(Alex_Layer5_pool);
	hipFree(Alex_Layer1_norm);
	hipFree(Alex_Layer2_norm);
	hipFree(Alex_Result_Neurons);

	hipFree(Res_Layer1_Neurons);
    hipFree(Res_Layer2_Neurons);
	hipFree(Res_Layer3_Neurons);
	hipFree(Res_Layer4_Neurons);
	hipFree(Res_Layer5_Neurons);
	hipFree(Res_Layer6_Neurons);
	hipFree(Res_Layer7_Neurons);
	hipFree(Res_Layer8_Neurons);
	hipFree(Res_Layer9_Neurons);
	hipFree(Res_Layer10_Neurons);
	hipFree(Res_Layer11_Neurons);
	hipFree(Res_Layer12_Neurons);
	hipFree(Res_Layer13_Neurons);
	hipFree(Res_Layer14_Neurons);
	hipFree(Res_Layer15_Neurons);
	hipFree(Res_Layer16_Neurons);
	hipFree(Res_Layer17_Neurons);
	hipFree(Res_Layer18_Neurons);

	hipFree(Res_Layer1_Weights);
	hipFree(Res_Layer2_Weights);
	hipFree(Res_Layer3_Weights);
	hipFree(Res_Layer4_Weights);
	hipFree(Res_Layer5_Weights);
	hipFree(Res_Layer6_Weights);
	hipFree(Res_Layer7_Weights);
	hipFree(Res_Layer8_Weights);
	hipFree(Res_Layer9_Weights);
	hipFree(Res_Layer10_Weights);
	hipFree(Res_Layer11_Weights);
	hipFree(Res_Layer12_Weights);
	hipFree(Res_Layer13_Weights);
	hipFree(Res_Layer14_Weights);
	hipFree(Res_Layer15_Weights);
	hipFree(Res_Layer16_Weights);
	hipFree(Res_Layer17_Weights);
	hipFree(Res_Block3_Weights);
	hipFree(Res_Block4_Weights);
	hipFree(Res_Block5_Weights);

	hipFree(Res_Layer1_Gamma);
	hipFree(Res_Layer2_Gamma);
	hipFree(Res_Layer3_Gamma);
	hipFree(Res_Layer4_Gamma);
	hipFree(Res_Layer5_Gamma);
	hipFree(Res_Layer6_Gamma);
	hipFree(Res_Layer7_Gamma);
	hipFree(Res_Layer8_Gamma);
	hipFree(Res_Layer9_Gamma);
	hipFree(Res_Layer10_Gamma);
	hipFree(Res_Layer11_Gamma);
	hipFree(Res_Layer12_Gamma);
	hipFree(Res_Layer13_Gamma);
	hipFree(Res_Layer14_Gamma);
	hipFree(Res_Layer15_Gamma);
	hipFree(Res_Layer16_Gamma);
	hipFree(Res_Layer17_Gamma);
	hipFree(Res_Block3_Gamma);
	hipFree(Res_Block4_Gamma);
	hipFree(Res_Block5_Gamma);

	hipFree(Res_Layer1_Beta);
	hipFree(Res_Layer2_Beta);
	hipFree(Res_Layer3_Beta);
	hipFree(Res_Layer4_Beta);
	hipFree(Res_Layer5_Beta);
	hipFree(Res_Layer6_Beta);
	hipFree(Res_Layer7_Beta);
	hipFree(Res_Layer8_Beta);
	hipFree(Res_Layer9_Beta);
	hipFree(Res_Layer10_Beta);
	hipFree(Res_Layer11_Beta);
	hipFree(Res_Layer12_Beta);
	hipFree(Res_Layer13_Beta);
	hipFree(Res_Layer14_Beta);
	hipFree(Res_Layer15_Beta);
	hipFree(Res_Layer16_Beta);
	hipFree(Res_Layer17_Beta);
	hipFree(Res_Block3_Beta);
	hipFree(Res_Block4_Beta);
	hipFree(Res_Block5_Beta);

	hipFree(Res_mean1);
	hipFree(Res_mean2);
	hipFree(Res_mean3);
	hipFree(Res_mean4);
	hipFree(Res_mean5);
	hipFree(Res_mean6);
	hipFree(Res_mean7);
	hipFree(Res_mean8);
	hipFree(Res_mean9);
	hipFree(Res_mean10);
	hipFree(Res_mean11);
	hipFree(Res_mean12);
	hipFree(Res_mean13);
	hipFree(Res_mean14);
	hipFree(Res_mean15);
	hipFree(Res_mean16);
	hipFree(Res_mean17);
	hipFree(Res_Block3_mean);
	hipFree(Res_Block4_mean);
	hipFree(Res_Block5_mean);

	hipFree(Res_var1);
	hipFree(Res_var2);
	hipFree(Res_var3);
	hipFree(Res_var4);
	hipFree(Res_var5);
	hipFree(Res_var6);
	hipFree(Res_var7);
	hipFree(Res_var8);
	hipFree(Res_var9);
	hipFree(Res_var10);
	hipFree(Res_var11);
	hipFree(Res_var12);
	hipFree(Res_var13);
	hipFree(Res_var14);
	hipFree(Res_var15);
	hipFree(Res_var16);
	hipFree(Res_var17);
	hipFree(Res_Block3_var);
	hipFree(Res_Block4_var);
	hipFree(Res_Block5_var);

	hipFree(Res_FC_bias);
	hipFree(Res_FC_Weights);

	hipFree(Res_Layer3_basic);
	hipFree(Res_Layer5_basic);
	hipFree(Res_Layer7_basic);
	hipFree(Res_Layer9_basic);
	hipFree(Res_Layer11_basic);
	hipFree(Res_Layer13_basic);
	hipFree(Res_Layer15_basic);
	hipFree(Res_Layer17_basic);
	hipFree(Res_Block3_basic);
	hipFree(Res_Block4_basic);
	hipFree(Res_Block5_basic);
	hipFree(Res_Layer1_bn);
	hipFree(Res_Layer2_bn);
	hipFree(Res_Layer3_bn);
	hipFree(Res_Layer4_bn);
	hipFree(Res_Layer5_bn);
	hipFree(Res_Layer6_bn);
	hipFree(Res_Layer7_bn);
	hipFree(Res_Layer8_bn);
	hipFree(Res_Layer9_bn);
	hipFree(Res_Layer10_bn);
	hipFree(Res_Layer11_bn);
	hipFree(Res_Layer12_bn);
	hipFree(Res_Layer13_bn);
	hipFree(Res_Layer14_bn);
	hipFree(Res_Layer15_bn);
	hipFree(Res_Layer16_bn);
	hipFree(Res_Layer17_bn);
	hipFree(Res_Block3_bn);
	hipFree(Res_Block4_bn);
	hipFree(Res_Block5_bn);
	hipFree(Res_Layer1_pool);
	hipFree(Res_FC_Neurons);
	hipFree(Res_Result_Neurons);
}
